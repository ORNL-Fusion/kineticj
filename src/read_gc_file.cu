#include "c3vec.hpp"
#include "cspecies.hpp"
#include <fstream>
#include <iostream>
#include <netcdf>
#include <string>
#include <vector>

using namespace netCDF;
using namespace netCDF::exceptions;

int read_gc_file(std::string fName,
    std::vector<float>& r_gc, std::vector<C3<float> >& curv_CYL, std::vector<C3<float> >& grad_CYL,
    std::vector<float>& bDotGradB)
{

    // Read the guiding center terms from file
    std::cout << "Reading GC terms data file " << fName << std::endl;

    std::vector<float> curv_r, curv_p, curv_z,
        grad_r, grad_p, grad_z;

    std::ifstream gc_file(fName.c_str());
    if (!gc_file.good()) {
        std::cout << "ERROR: Cannot find file " << fName << std::endl;
        return (1);
    }

    NcFile dataFile(fName.c_str(), NcFile::read);
    NcDim gc_nc_nR(dataFile.getDim("nR"));
    NcDim gc_nc_scalar(dataFile.getDim("scalar"));
    if (!dataFile.getVar("z").isNull())
        throw NcException("This is a 2D GC terms file", __FILE__, __LINE__);

    int nR_gc = gc_nc_nR.getSize();
    std::cout << "nR_gc: " << nR_gc << std::endl;
    NcVar gc_nc_r(dataFile.getVar("r"));
    NcVar gc_nc_curv_r(dataFile.getVar("curv_r"));
    NcVar gc_nc_curv_p(dataFile.getVar("curv_t"));
    NcVar gc_nc_curv_z(dataFile.getVar("curv_z"));

    NcVar gc_nc_grad_r(dataFile.getVar("grad_r"));
    NcVar gc_nc_grad_p(dataFile.getVar("grad_t"));
    NcVar gc_nc_grad_z(dataFile.getVar("grad_z"));

    NcVar gc_nc_bDotGradB(dataFile.getVar("bDotGradB"));

    r_gc.resize(nR_gc);

    curv_r.resize(nR_gc);
    curv_p.resize(nR_gc);
    curv_z.resize(nR_gc);

    grad_r.resize(nR_gc);
    grad_p.resize(nR_gc);
    grad_z.resize(nR_gc);

    bDotGradB.resize(nR_gc);

    gc_nc_r.getVar(&r_gc[0]);

    gc_nc_curv_r.getVar(&curv_r[0]);
    gc_nc_curv_p.getVar(&curv_p[0]);
    gc_nc_curv_z.getVar(&curv_z[0]);

    gc_nc_grad_r.getVar(&grad_r[0]);
    gc_nc_grad_p.getVar(&grad_p[0]);
    gc_nc_grad_z.getVar(&grad_z[0]);

    gc_nc_bDotGradB.getVar(&bDotGradB[0]);

    curv_CYL.resize(nR_gc);
    grad_CYL.resize(nR_gc);
    for (int i = 0; i < nR_gc; i++) {
        curv_CYL[i] = C3<float>(curv_r[i], curv_p[i], curv_z[i]);
        grad_CYL[i] = C3<float>(grad_r[i], grad_p[i], grad_z[i]);
    }
    std::cout << "Finished reading gc_terms file" << std::endl;

    return (0);
}
