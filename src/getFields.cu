#include "hip/hip_runtime.h"
#include "getFields.hpp"

#ifdef __HIPCC__
HOST DEVICE
C3<thrust::complex<float> > getE1orB1_XYZ_fromCYL(CParticle& p_XYZ, float *rVec, C3<thrust::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<thrust::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    thrust::complex<float> ii(0, 1);

    E1_XYZ = thrust::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}
#endif

HOST
C3<std::complex<float> > getE1orB1_XYZ_fromCYL(CParticle& p_XYZ, float *rVec, C3<std::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<std::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    std::complex<float> ii(0, 1);

    E1_XYZ = std::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}

#ifdef __HIPCC__
HOST DEVICE
C3<thrust::complex<float> > getE1orB1_XYZ_fromXYZ(CParticle& p_XYZ, float *rVec, C3<thrust::complex<float> > *E1Vec_CYL, int nR, float ky)
{

    float _x = p_XYZ.c1;
    float _y = p_XYZ.c2;
    float _z = p_XYZ.c3;

    C3<thrust::complex<float> > E1_XYZ;

    int status = 0;
    thrust::complex<float> ii(0, 1);

    E1_XYZ = thrust::exp(ii * float(ky*_y)) * kj_interp1D(_x, rVec, E1Vec_CYL, nR, status);

    p_XYZ.status = max(p_XYZ.status, status);

    return E1_XYZ;
}
#endif

HOST
C3<std::complex<float> > getE1orB1_XYZ_fromXYZ(CParticle& p_XYZ, float *rVec, C3<std::complex<float> > *E1Vec_CYL, int nR, float ky)
{

    float _x = p_XYZ.c1;
    float _y = p_XYZ.c2;
    float _z = p_XYZ.c3;

    C3<std::complex<float> > E1_XYZ;

    int status = 0;
    std::complex<float> ii(0, 1);

    E1_XYZ = std::exp(ii * float(ky * _y)) * kj_interp1D(_x, rVec, E1Vec_CYL, nR, status);

    p_XYZ.status = max(p_XYZ.status, status);

    return E1_XYZ;
}


