#include "hip/hip_runtime.h"
#include "getFields.hpp"
#include <iomanip>

#if defined(__HIPCC__) || defined(__THRUST)
HOST DEVICE
C3<thrust::complex<float> > getE1orB1_XYZ_fromCYL(CParticle& p_XYZ, float *rVec, C3<thrust::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<thrust::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    thrust::complex<float> ii(0, 1);

    E1_XYZ = thrust::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}
#endif

HOST
C3<std::complex<float> > getE1orB1_XYZ_fromCYL(CParticle& p_XYZ, float *rVec, C3<std::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<std::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    std::complex<float> ii(0, 1);

    E1_XYZ = std::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}

#if defined(__HIPCC__) || defined(__THRUST)
HOST DEVICE
C3<thrust::complex<float> > getE1orB1_XYZ_fromXYZ(CParticle& p_XYZ, float *rVec, C3<thrust::complex<float> > *E1Vec_CYL, int nR, float ky, float kz)
{

    float _x = p_XYZ.c1;
    float _y = p_XYZ.c2;
    float _z = p_XYZ.c3;

    C3<thrust::complex<float> > E1_XYZ;

    int status = 0;
    thrust::complex<float> ii(0, 1);

    //float kx;
    //kx = 10.0;

    E1_XYZ = thrust::exp( ii * ( ky * _y + kz * _z ) ) * kj_interp1D(_x, rVec, E1Vec_CYL, nR, status);
    //E1_XYZ.c2 = 0.0;
    //E1_XYZ.c3 = 0.0;

    p_XYZ.status = max(p_XYZ.status, status);

    return E1_XYZ;
}
#endif

HOST
C3<std::complex<float> > getE1orB1_XYZ_fromXYZ(CParticle& p_XYZ, float *rVec, C3<std::complex<float> > *E1Vec_CYL, int nR, float ky, float kz)
{

    float _x = p_XYZ.c1;
    float _y = p_XYZ.c2;
    float _z = p_XYZ.c3;

    C3<std::complex<float> > E1_XYZ;

    int status = 0;
    std::complex<float> ii(0, 1);

    //float kx;
    //kx = 10.0;

    E1_XYZ = std::exp(ii * ( ky * _y + kz * _z ) ) * kj_interp1D(_x, rVec, E1Vec_CYL, nR, status);
    //std::cout<<std::setprecision(21)<<"_x: "<<_x<<" _y: "<<_y<<" _z: "<<_z<<" kx: "<<kx<<" E: "<<E1_XYZ.c1<<std::endl;
    //E1_XYZ.c2 = 0.0;
    //E1_XYZ.c3 = 0.0;

    p_XYZ.status = max(p_XYZ.status, status);

    return E1_XYZ;
}


