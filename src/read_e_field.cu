#include "hip/hip_runtime.h"
#include "c3vec.hpp"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <netcdf>
#include <new> // std::bad_alloc
#include <string>
#include <vector>
#include <cmath>

using namespace netCDF;
using namespace exceptions;

int read_e_field(std::string eField_fName, int& species_number, float& freq,
    std::vector<float>& r, std::vector<float>& n_m3,
    std::vector<C3<std::complex<float> > >& e1_CYL, 
    std::vector<C3<std::complex<float> > >& b1_CYL,
    std::vector<C3<float> >& b0_CYL)
{

    std::cout << "Reading eField data file " << eField_fName << std::endl;

    // Here we are using the cxx-4 netcdf interface by Lynton Appel
    // This needs netCDF 4.1.1 or later build with
    // ./configure --enable-cxx-4 [plus other options]

    std::vector<float> b0_r, b0_p, b0_z,
        e_r_re, e_p_re, e_z_re,
        e_r_im, e_p_im, e_z_im,
        b_r_re, b_p_re, b_z_re,
        b_r_im, b_p_im, b_z_im;

    std::vector<std::complex<float> > e_r, e_p, e_z;
    std::vector<std::complex<float> > b_r, b_p, b_z;

    std::ifstream file(eField_fName.c_str());
    if (!file.good()) {
        std::cout << "ERROR: Cannot find file " << eField_fName << std::endl;
        exit(1);
    }

    try {
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Message: DEBUG_READ_E_FIELD enabled"<<std::endl;
#endif
        std::cout << "Reading E field data file ... " << eField_fName << std::endl;
        NcFile dataFile(eField_fName.c_str(), NcFile::read);

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Pre getDim"<<std::endl;
#endif
        NcDim nc_nR(dataFile.getDim("nR"));
        NcDim nc_nSpec(dataFile.getDim("nSpec"));
        NcDim nc_scalar(dataFile.getDim("scalar"));

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Post getDim "<<std::endl;
#endif
 
        int nR = nc_nR.getSize();
        int nSpec = nc_nSpec.getSize();

        if (species_number > nSpec - 1) {
            std::cout << "ERROR: Asking for species that does not exist in density data" << std::endl;
            exit(1);
        }
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Post spec num check "<<std::endl;
#endif
        std::cout << "\tnR: " << nR << std::endl;

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Pre getVar "<<std::endl;
#endif
        NcVar nc_r(dataFile.getVar("r"));
        NcVar nc_freq(dataFile.getVar("freq"));

        NcVar nc_b0_r(dataFile.getVar("B0_r"));
        NcVar nc_b0_p(dataFile.getVar("B0_p"));
        NcVar nc_b0_z(dataFile.getVar("B0_z"));

        NcVar nc_e_r_re(dataFile.getVar("e_r_re"));
        NcVar nc_e_p_re(dataFile.getVar("e_p_re"));
        NcVar nc_e_z_re(dataFile.getVar("e_z_re"));
        NcVar nc_e_r_im(dataFile.getVar("e_r_im"));
        NcVar nc_e_p_im(dataFile.getVar("e_p_im"));
        NcVar nc_e_z_im(dataFile.getVar("e_z_im"));

        NcVar nc_b_r_re(dataFile.getVar("b_r_re"));
        NcVar nc_b_p_re(dataFile.getVar("b_p_re"));
        NcVar nc_b_z_re(dataFile.getVar("b_z_re"));
        NcVar nc_b_r_im(dataFile.getVar("b_r_im"));
        NcVar nc_b_p_im(dataFile.getVar("b_p_im"));
        NcVar nc_b_z_im(dataFile.getVar("b_z_im"));

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Pre getVar density"<<std::endl;
#endif
        NcVar nc_density(dataFile.getVar("density_m3"));

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Post getVar density"<<std::endl;
#endif
        r.resize(nR);

        b0_r.resize(nR);
        b0_p.resize(nR);
        b0_z.resize(nR);

        e_r_re.resize(nR);
        e_p_re.resize(nR);
        e_z_re.resize(nR);
        e_r_im.resize(nR);
        e_p_im.resize(nR);
        e_z_im.resize(nR);

        b_r_re.resize(nR);
        b_p_re.resize(nR);
        b_z_re.resize(nR);
        b_r_im.resize(nR);
        b_p_im.resize(nR);
        b_z_im.resize(nR);

        n_m3.resize(nR);

        nc_r.getVar(&r[0]);
        nc_freq.getVar(&freq);

        nc_b0_r.getVar(&b0_r[0]);
        nc_b0_p.getVar(&b0_p[0]);
        nc_b0_z.getVar(&b0_z[0]);

        // Here im reading a single species' density from a multi species array,
        // i.e., density[nSpec,nR] and I only want density[1,*] for example where
        // the species is specified by "species_number" in the cfg file
        std::vector<size_t> start, count;
        start.resize(2);
        count.resize(2);
        start[0] = species_number;
        start[1] = 0;
        count[0] = 1;
        count[1] = nR;

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Pre nc_density.getVar()"<<std::endl;
#endif
        nc_density.getVar(start, count, &n_m3[0]);
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Post nc_density.getVar()"<<std::endl;
#endif
 
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"nR: "<<nR<<std::endl;
        for (int i = 0; i < nR; i++) {
            std::cout<<"density: "<<n_m3[i]<<std::endl;
        }
#endif
        try {
            std::cout << "nR : " << nR << std::endl;
            b0_CYL.resize(nR);
        } catch (const std::bad_alloc& error) {
            std::cout << "Allocation error at " << __FILE__ << __LINE__ << std::endl;
            std::cout << error.what();
        }

        for (int i = 0; i < nR; i++) {
            b0_CYL[i] = C3<float>(b0_r[i], b0_p[i], b0_z[i]);
        }

#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Pre nc_e_r_re.getVar()"<<std::endl;
#endif
        nc_e_r_re.getVar(&e_r_re[0]);
        nc_e_p_re.getVar(&e_p_re[0]);
        nc_e_z_re.getVar(&e_z_re[0]);
        nc_e_r_im.getVar(&e_r_im[0]);
        nc_e_p_im.getVar(&e_p_im[0]);
        nc_e_z_im.getVar(&e_z_im[0]);

        nc_b_r_re.getVar(&b_r_re[0]);
        nc_b_p_re.getVar(&b_p_re[0]);
        nc_b_z_re.getVar(&b_z_re[0]);
        nc_b_r_im.getVar(&b_r_im[0]);
        nc_b_p_im.getVar(&b_p_im[0]);
        nc_b_z_im.getVar(&b_z_im[0]);

        for (int i = 0; i < nR; i++) {
            e_r.push_back(std::complex<float>(e_r_re[i], e_r_im[i]));
            e_p.push_back(std::complex<float>(e_p_re[i], e_p_im[i]));
            e_z.push_back(std::complex<float>(e_z_re[i], e_z_im[i]));
        }

        for (int i = 0; i < nR; i++) {
            b_r.push_back(std::complex<float>(b_r_re[i], b_r_im[i]));
            b_p.push_back(std::complex<float>(b_p_re[i], b_p_im[i]));
            b_z.push_back(std::complex<float>(b_z_re[i], b_z_im[i]));
        }

        std::vector<float>::iterator min = std::min_element(b0_p.begin(), b0_p.end());
        std::vector<float>::iterator max = std::max_element(b0_p.begin(), b0_p.end());
#if DEBUGLEVEL >= 1
        std::cout << "\tR[0]: " << r[0] << ", R[" << nR << "]: " << r[r.size() - 1] << std::endl;
        std::cout << "\tfreq: " << freq << std::endl;
        std::cout << "\tmin(b0_p): " << *min << std::endl;
        std::cout << "\tmax(b0_p): " << *max << std::endl;
        std::cout << "\tabs(e_r[nR/2]): " << std::abs(e_r[nR / 2]) << std::endl;
        std::cout << "\tabs(e_p[nR/2]): " << std::abs(e_p[nR / 2]) << std::endl;
        std::cout << "\tabs(e_z[nR/2]): " << std::abs(e_z[nR / 2]) << std::endl;
#endif
    } catch (exceptions::NcException& e) {
        std::cout << "NetCDF: unknown error" << std::endl;
        e.what();
        exit(1);
    }

    e1_CYL.resize(e_r.size());
    b1_CYL.resize(b_r.size());

    for (int i = 0; i < e_r.size(); i++) {

        e1_CYL[i].c1 = e_r[i];
        e1_CYL[i].c2 = e_p[i]*std::complex<float>(0);
        e1_CYL[i].c3 = e_z[i]*std::complex<float>(0);

        b1_CYL[i].c1 = b_r[i];
        b1_CYL[i].c2 = b_p[i];
        b1_CYL[i].c3 = b_z[i];
    }

    std::cout << "End of " << __FILE__ << std::endl;
    return (0);
}
