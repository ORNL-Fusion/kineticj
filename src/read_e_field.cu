#include "hip/hip_runtime.h"
#include "c3vec.hpp"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <netcdf>
#include <new> // std::bad_alloc
#include <string>
#include <vector>
#include <cmath>

using namespace netCDF;
using namespace exceptions;

int read_e_field(std::string eField_fName, int& species_number, float& freq,
    std::vector<float>& r, std::vector<float>& n_m3,
    std::vector<C3<std::complex<float> > >& e1_CYL, std::vector<C3<std::complex<float> > >& b1_CYL,
    std::vector<C3<float> >& e1Re_CYL, std::vector<C3<float> >& e1Im_CYL,
    std::vector<C3<float> >& b1Re_CYL, std::vector<C3<float> >& b1Im_CYL,
    std::vector<C3<float> >& b0_CYL)
{

    std::cout << "Reading eField data file " << eField_fName << std::endl;

    // Here we are using the cxx-4 netcdf interface by Lynton Appel
    // This needs netCDF 4.1.1 or later build with
    // ./configure --enable-cxx-4 [plus other options]

    std::vector<float> b0_r, b0_p, b0_z,
        e_r_re, e_p_re, e_z_re,
        e_r_im, e_p_im, e_z_im,
        b_r_re, b_p_re, b_z_re,
        b_r_im, b_p_im, b_z_im;

    std::vector<std::complex<float> > e_r, e_p, e_z;
    std::vector<std::complex<float> > b_r, b_p, b_z;

    std::ifstream file(eField_fName.c_str());
    if (!file.good()) {
        std::cout << "ERROR: Cannot find file " << eField_fName << std::endl;
        exit(1);
    }

    try {
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"Message: DEBUG_READ_E_FIELD enabled"<<nR<<std::endl;
#endif
 
        std::cout << "Reading E field data file ... " << eField_fName << std::endl;
        NcFile dataFile(eField_fName.c_str(), NcFile::read);

        NcDim nc_nR(dataFile.getDim("nR"));
        NcDim nc_nSpec(dataFile.getDim("nSpec"));
        NcDim nc_scalar(dataFile.getDim("scalar"));

        int nR = nc_nR.getSize();
        int nSpec = nc_nSpec.getSize();

        if (species_number > nSpec - 1) {
            std::cout << "ERROR: Asking for species that does not exist in density data" << std::endl;
            exit(1);
        }

        std::cout << "\tnR: " << nR << std::endl;

        NcVar nc_r(dataFile.getVar("r"));
        NcVar nc_freq(dataFile.getVar("freq"));

        NcVar nc_b0_r(dataFile.getVar("B0_r"));
        NcVar nc_b0_p(dataFile.getVar("B0_p"));
        NcVar nc_b0_z(dataFile.getVar("B0_z"));

        NcVar nc_e_r_re(dataFile.getVar("e_r_re"));
        NcVar nc_e_p_re(dataFile.getVar("e_p_re"));
        NcVar nc_e_z_re(dataFile.getVar("e_z_re"));
        NcVar nc_e_r_im(dataFile.getVar("e_r_im"));
        NcVar nc_e_p_im(dataFile.getVar("e_p_im"));
        NcVar nc_e_z_im(dataFile.getVar("e_z_im"));

        NcVar nc_b_r_re(dataFile.getVar("b_r_re"));
        NcVar nc_b_p_re(dataFile.getVar("b_p_re"));
        NcVar nc_b_z_re(dataFile.getVar("b_z_re"));
        NcVar nc_b_r_im(dataFile.getVar("b_r_im"));
        NcVar nc_b_p_im(dataFile.getVar("b_p_im"));
        NcVar nc_b_z_im(dataFile.getVar("b_z_im"));

        NcVar nc_density(dataFile.getVar("density_m3"));

        r.resize(nR);

        b0_r.resize(nR);
        b0_p.resize(nR);
        b0_z.resize(nR);

        e_r_re.resize(nR);
        e_p_re.resize(nR);
        e_z_re.resize(nR);
        e_r_im.resize(nR);
        e_p_im.resize(nR);
        e_z_im.resize(nR);

        b_r_re.resize(nR);
        b_p_re.resize(nR);
        b_z_re.resize(nR);
        b_r_im.resize(nR);
        b_p_im.resize(nR);
        b_z_im.resize(nR);

        n_m3.resize(nR);

        nc_r.getVar(&r[0]);
        nc_freq.getVar(&freq);

        nc_b0_r.getVar(&b0_r[0]);
        nc_b0_p.getVar(&b0_p[0]);
        nc_b0_z.getVar(&b0_z[0]);

        // Here im reading a single species' density from a multi species array,
        // i.e., density[nSpec,nR] and I only want density[1,*] for example where
        // the species is specified by "species_number" in the cfg file
        std::vector<size_t> start, count;
        start.resize(2);
        count.resize(2);
        start[1] = 0;
        start[0] = species_number;
        count[1] = nR;
        count[0] = 1;

        nc_density.getVar(start, count, &n_m3[0]);
#if DEBUG_READ_E_FIELD >= 1
        std::cout<<"nR: "<<nR<<std::endl;
        for (int i = 0; i < nR; i++) {
            std::cout<<"density: "<<nc_density[i]<<std::endl;
        }
#endif
        try {
            std::cout << "nR : " << nR << std::endl;
            b0_CYL.resize(nR);
        } catch (const std::bad_alloc& error) {
            std::cout << "Allocation error at " << __FILE__ << __LINE__ << std::endl;
            std::cout << error.what();
        }

        for (int i = 0; i < nR; i++) {
            b0_CYL[i] = C3<float>(b0_r[i], b0_p[i], b0_z[i]);
        }

        nc_e_r_re.getVar(&e_r_re[0]);
        nc_e_p_re.getVar(&e_p_re[0]);
        nc_e_z_re.getVar(&e_z_re[0]);
        nc_e_r_im.getVar(&e_r_im[0]);
        nc_e_p_im.getVar(&e_p_im[0]);
        nc_e_z_im.getVar(&e_z_im[0]);

        nc_b_r_re.getVar(&b_r_re[0]);
        nc_b_p_re.getVar(&b_p_re[0]);
        nc_b_z_re.getVar(&b_z_re[0]);
        nc_b_r_im.getVar(&b_r_im[0]);
        nc_b_p_im.getVar(&b_p_im[0]);
        nc_b_z_im.getVar(&b_z_im[0]);

        for (int i = 0; i < nR; i++) {
            e_r.push_back(std::complex<float>(e_r_re[i], e_r_im[i]));
            e_p.push_back(std::complex<float>(e_p_re[i], e_p_im[i]));
            e_z.push_back(std::complex<float>(e_z_re[i], e_z_im[i]));
        }

        for (int i = 0; i < nR; i++) {
            b_r.push_back(std::complex<float>(b_r_re[i], b_r_im[i]));
            b_p.push_back(std::complex<float>(b_p_re[i], b_p_im[i]));
            b_z.push_back(std::complex<float>(b_z_re[i], b_z_im[i]));
        }

        std::vector<float>::iterator min = std::min_element(b0_p.begin(), b0_p.end());
        std::vector<float>::iterator max = std::max_element(b0_p.begin(), b0_p.end());
#if DEBUGLEVEL >= 1
        std::cout << "\tR[0]: " << r[0] << ", R[" << nR << "]: " << r[r.size() - 1] << std::endl;
        std::cout << "\tfreq: " << freq << std::endl;
        std::cout << "\tmin(b0_p): " << *min << std::endl;
        std::cout << "\tmax(b0_p): " << *max << std::endl;
        std::cout << "\tabs(e_r[nR/2]): " << std::abs(e_r[nR / 2]) << std::endl;
        std::cout << "\tabs(e_p[nR/2]): " << std::abs(e_p[nR / 2]) << std::endl;
        std::cout << "\tabs(e_z[nR/2]): " << std::abs(e_z[nR / 2]) << std::endl;
#endif
    } catch (exceptions::NcException& e) {
        std::cout << "NetCDF: unknown error" << std::endl;
        e.what();
        exit(1);
    }

    e1Re_CYL.resize(e_r.size());
    e1Im_CYL.resize(e_r.size());
    b1Re_CYL.resize(e_r.size());
    b1Im_CYL.resize(e_r.size());

    e1_CYL.resize(e_r.size());
    b1_CYL.resize(b_r.size());

    for (int i = 0; i < e_r.size(); i++) {

        e1Re_CYL[i].c1 = real(e_r[i]);
        e1Re_CYL[i].c2 = real(e_p[i]);
        e1Re_CYL[i].c3 = real(e_z[i]);
        e1Im_CYL[i].c1 = imag(e_r[i]);
        e1Im_CYL[i].c2 = imag(e_p[i]);
        e1Im_CYL[i].c3 = imag(e_z[i]);

        b1Re_CYL[i].c1 = real(b_r[i]);
        b1Re_CYL[i].c2 = real(b_p[i]);
        b1Re_CYL[i].c3 = real(b_z[i]);
        b1Im_CYL[i].c1 = imag(b_r[i]);
        b1Im_CYL[i].c2 = imag(b_p[i]);
        b1Im_CYL[i].c3 = imag(b_z[i]);

        e1_CYL[i].c1 = e_r[i];
        e1_CYL[i].c2 = e_p[i];
        e1_CYL[i].c3 = e_z[i];

        b1_CYL[i].c1 = b_r[i];
        b1_CYL[i].c2 = b_p[i];
        b1_CYL[i].c3 = b_z[i];
    }

    std::cout << "End of " << __FILE__ << std::endl;
    return (0);
}
