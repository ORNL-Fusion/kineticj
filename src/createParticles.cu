#include "hip/hip_runtime.h"
#include "createParticles.hpp"
#include "rk4.hpp"
#include <cmath>

float GetGyroPhase(const C3<float> v_abp)
{

    // alp is mostly in the x / r direction
    // bet is mostly z direction

    float alp = v_abp.c1;
    float bet = v_abp.c2;

    return atan2(alp, bet);
}

PRAGMA
HOST DEVICE
float maxwellian(float vx, float vy, float vz, float vTh)
{

    float weight_x = 1.0 / (vTh * std::sqrt(physConstants::pi)) * std::exp(-std::pow(vx, 2) / std::pow(vTh, 2));
    float weight_y = 1.0 / (vTh * std::sqrt(physConstants::pi)) * std::exp(-std::pow(vy, 2) / std::pow(vTh, 2));
    float weight_z = 1.0 / (vTh * std::sqrt(physConstants::pi)) * std::exp(-std::pow(vz, 2) / std::pow(vTh, 2));

    return weight_x * weight_y * weight_z;
}

PRAGMA
HOST DEVICE
float get_vTh(const float _amu, const float _Z, const float _T_keV)
{

    float m = _amu * physConstants::amu;
    float kT_joule = _T_keV * 1e3 * physConstants::e; // This may actually be E_keV so may need a 3/2 somewhere
    float vTh = std::sqrt(2.0 * kT_joule / m);

    return vTh;
}

PRAGMA
HOST DEVICE
C3<float> maxwellian_df0_dv(const C3<float> _v, const float _T_keV, const float _n_m3, const float _amu, const float _Z)
{

    C3<float> df0_dv;

    float vTh = get_vTh(_amu, _Z, _T_keV);

    float _vx = _v.c1;
    float _vy = _v.c2;
    float _vz = _v.c3;

    // Get the 3 components of df0_dv at this point in velocity space

    float h = vTh / 1000.0;
    float vxL = _vx - h;
    float vxR = _vx + h;
    float fL = maxwellian(vxL, _vy, _vz, vTh);
    float fR = maxwellian(vxR, _vy, _vz, vTh);
    float _df0_dv = (-fL + fR) / (2 * h);

    df0_dv.c1 = _df0_dv * _n_m3;

    float vyL = _vy - h;
    float vyR = _vy + h;
    fL = maxwellian(_vx, vyL, _vz, vTh);
    fR = maxwellian(_vx, vyR, _vz, vTh);
    _df0_dv = (-fL + fR) / (2 * h);

    df0_dv.c2 = _df0_dv * _n_m3;

    float vzL = _vz - h;
    float vzR = _vz + h;
    fL = maxwellian(_vx, _vy, vzL, vTh);
    fR = maxwellian(_vx, _vy, vzR, vTh);
    _df0_dv = (-fL + fR) / (2 * h);

    df0_dv.c3 = _df0_dv * _n_m3;

    return df0_dv;
}

vector<CParticle> create_particles(float x, float amu, float Z, float T_keV, float n_m3,
    int nPx, int nPy, int nPz, int nThermal, float& dv, float *r, C3<float> *b0_CYL, int nR)
{

    vector<CParticle> pList;

    int nP = nPx * nPy * nPz;
    pList.resize(nP);

    float vTh = get_vTh(amu, Z, T_keV);

#if DEBUG_MAXWELLIAN >= 1
    cout << "amu: " << amu << endl;
    cout << "Z: " << Z << endl;
    cout << "vTh: " << vTh << endl;
#endif

    float vxRange = vTh * nThermal * 2;
    float vxMin = -vxRange / 2.0;
    float dvx = vxRange / (nPx - 1);

    float vyRange = vTh * nThermal * 2;
    float vyMin = -vyRange / 2.0;
    float dvy = vyRange / (nPy - 1);

    float vzRange = vTh * nThermal * 2;
    float vzMin = -vzRange / 2.0;
    float dvz = vzRange / (nPz - 1);

    dv = dvx * dvy * dvz; // Return the Jacobian (volume element for integration later)

    float TestIntegratedValue = 0;

    int cnt = 0;
    for (int i = 0; i < nPx; i++) {
        for (int j = 0; j < nPy; j++) {
            for (int k = 0; k < nPz; k++) {

                float thisvx = vxMin + i * dvx;
                float thisvy = vyMin + j * dvy;
                float thisvz = vzMin + k * dvz;

                float weight = maxwellian(thisvx, thisvy, thisvz, vTh) * n_m3;

                TestIntegratedValue += weight * dv;

                CParticle p(x, 0.0, 0.0, thisvx, thisvy, thisvz, amu, Z, weight, T_keV, n_m3);
                pList[cnt] = p;
                pList[cnt].number = cnt;
                pList[cnt].vTh = vTh;

                pList[cnt].d3v = dv;

                // Get vPar, vPer and mu for guiding center integration

                C3<float> thisV_XYZ(thisvx, thisvy, thisvz);
                int iStat = 0;
                C3<float> this_b0_CYL = kj_interp1D(x, r, b0_CYL, nR, iStat);
                if(iStat>0) {
                    cout << "ERROR : Interpolation failure on b0_CYL" << endl;
                    exit(1);
                }
                C3<float> this_b0_XYZ = rot_CYL_to_XYZ(0, this_b0_CYL, 1);
                float bMag = mag(this_b0_XYZ);
                float vMag = mag(thisV_XYZ);

                C3<float> thisV_abp = rot_XYZ_to_abp(thisV_XYZ, this_b0_XYZ, 0);

                pList[cnt].vPar = thisV_abp.c3;
                pList[cnt].vPer = std::sqrt(std::pow(thisV_abp.c1, 2) + std::pow(thisV_abp.c2, 2));
                pList[cnt].gyroPhase = GetGyroPhase(thisV_abp);
                pList[cnt].u = pList[cnt].m * std::pow(pList[cnt].vPer, 2) / (2.0 * bMag);

#if GC_ORBITS > 0
                // Update the starting point to be at the guiding center

                int nTGC = 40;
                float wc = std::abs(pList[cnt].q * bMag / pList[cnt].m);
                float dTGC = 2*physConstants::pi/wc/nTGC;
                CParticle pGC(pList[cnt]);
                float averageX=0;
                float averageY=0;
                float averageZ=0;
                int MoveStatus = 0;
                for(int iGC=0; iGC<nTGC; iGC++) {
                    MoveStatus = rk4_move(pGC, dTGC, r, b0_CYL, nR);

                    averageX += pGC.c1;
                    averageY += pGC.c2;
                    averageZ += pGC.c3;
                }
                averageX = averageX/nTGC;
                averageY = averageY/nTGC;
                averageZ = averageZ/nTGC;

                pList[cnt].c1 = averageX;
                pList[cnt].c2 = averageY;
                pList[cnt].c3 = averageZ;
#endif

#if DEBUG_MAXWELLIAN >= 2
                cout << "ThisVx: " << thisvx << endl;
                cout << "ThisVy: " << thisvy << endl;
                cout << "ThisVz: " << thisvz << endl;
                cout << "b0_XYZ: " << this_b0_XYZ.c1 << ", " << this_b0_XYZ.c2 << ", " << this_b0_XYZ.c3 << endl;
                cout << "vMag: " << vMag << endl;
                cout << "vPer: " << pList[cnt].vPer << endl;
                cout << "vPar: " << pList[cnt].vPar << endl;
                cout << "u: " << pList[cnt].u << endl
                     << endl;
                if (isnan(pList[cnt].u))
                    exit(1);
                if (vMag > 3e8)
                    exit(1);
#endif
                cnt++;
            }
        }
    }

#if DEBUG_MAXWELLIAN >= 1
    cout << "TestIntegratedValue: " << TestIntegratedValue << endl;
#endif
    return pList;
}
