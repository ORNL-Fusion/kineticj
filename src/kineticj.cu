#include "hip/hip_runtime.h"
#include "c3vec.hpp"
#include "constants.hpp"
#include "cparticle.hpp"
#include "createParticles.hpp"
#include "cspecies.hpp"
#include "interp.hpp"
#include "read_e_field.hpp"
#include "read_gc_file.hpp"
#include "rk4.hpp"
#include "rotation.hpp"
#include <algorithm>
#include <assert.h>
#include <complex>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <libconfig.h++>
#include <netcdf>
#include <new> // for stl::bad_alloc
#include <omp.h>
#include <string>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <vector>
#include <numeric>

#ifdef __HIPCC__
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#endif

#if CLOCK >= 1
#include <ctime>
#endif

#if USEPAPI >= 1
#include <papi.h>
#endif

#if LOWMEM_USEPAPI >= 1
#include <papi.h>
#endif

//#include <google/profiler.h>

#ifdef __CUDA_ARCH__
#define PRINT cuPrintf
#else
#define PRINT printf
#endif

using namespace netCDF;
using namespace exceptions;

// Calculate the jP given some know E and f(v)

int main(int argc, char** argv)
{

#ifdef __HIPCC__

    int num_gpus = 0;   // number of CUDA GPUs

    printf("%s Starting...\n\n", argv[0]);

    // determine the number of CUDA capable GPUs
    hipGetDeviceCount(&num_gpus);

    if (num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }

    // display CPU and GPU configuration
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);

    for (int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }

#endif

    // Make sure the "output/" directory exists

    stringstream outputDirName;
    outputDirName << "output/";

    // check directory exists
    struct stat st;
    int dirTest = stat(outputDirName.str().c_str(), &st);
    if (dirTest != 0) {
        std::cout << "Had to create output/ directory" << std::endl;
        int mkDirStat = mkdir(outputDirName.str().c_str(),
            S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    }

#if CLOCK >= 1
    clock_t ProgramTime = clock();
#endif

#if (USEPAPI >= 1 || LOWMEM_USEPAPI >= 1)
    float realTime0, cpuTime0, realTime = 0, cpuTime = 0, mFlops = 0;
    long long flpIns0, flpIns = 0;
    int papiReturn;

    cpuTime0 = cpuTime;
    realTime0 = realTime;
    flpIns0 = flpIns;
    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    if (papiReturn < 0) {
        std::cout << "ERROR: PAPI Failed to initialize with error code: " << papiReturn
             << std::endl;
        std::cout << "ERROR: See papi.h for error code explanations " << std::endl;
        exit(1);
    }
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime, cpuTime, flpIns, mFlops);

    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    if (papiReturn < 0) {
        std::cout << "ERROR: PAPI Failed to initialize with error code: " << papiReturn
             << std::endl;
        std::cout << "ERROR: See papi.h for error code explanations " << std::endl;
        exit(1);
    } else {
        std::cout << "PAPI called successfully with return code: " << papiReturn << std::endl;
    }
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime, cpuTime, flpIns, mFlops);
#endif

    // Read config file

    libconfig::Config cfg;
    string cfgName = "kj.cfg";

    try {
        cfg.readFile(cfgName.c_str());
    } catch (const libconfig::FileIOException& fioex) {
        std::cerr << "I/O error while reading file." << std::endl;
        return (EXIT_FAILURE);
    } catch (const libconfig::ParseException& pex) {
        std::cerr << "Parse error at " << pex.getFile() << ":" << pex.getLine()
                  << " - " << pex.getError() << std::endl;
        return (EXIT_FAILURE);
    }

    int species_number = cfg.lookup("species_number");
    float T_keV_cfg = cfg.lookup("T_keV");

    // Read E
    string input_fName = cfg.lookup("input_fName");
    vector<C3<std::complex<float> > > e1_CYL, b1_CYL;
    vector<C3<float> > b0_CYL, b0_XYZ;
    vector<float> r, n_m3;
    float freq;
    int eReadStat = read_e_field(input_fName, species_number, freq, r, n_m3, 
                    e1_CYL, b1_CYL, b0_CYL);
#if GC_ORBITS >= 1

    // Read GC terms
    std::string gc_fName;
    if(cfg.lookupValue("gc_fName",gc_fName)) {
    } else {
        gc_fName = "gc_terms.nc";
    }
    //string gc_fName = cfg.lookup("gc_fName");

    vector<C3<float> > curv_CYL, grad_CYL;
    std::vector<float> r_gc, bDotGradB;
    int gcReadStat = read_gc_file(gc_fName, r_gc, curv_CYL, grad_CYL, bDotGradB);

#endif

    float wrf = freq * 2 * physConstants::pi;
    float xGridMin = cfg.lookup("xGridMin");
    float xGridMax = cfg.lookup("xGridMax");
    int nXGrid = cfg.lookup("nXGrid");
    std::cout << "nXGrid: " << nXGrid << std::endl;

    vector<float> xGrid(nXGrid);
    vector<float> density_m3(nXGrid);
    vector<float> T_keV(nXGrid);
    vector<float> wrf_wc(nXGrid);
    vector<float> bMag_kjGrid(nXGrid);

    float xGridRng = 0;
    float xGridStep = 0;

    if (nXGrid > 1) {
        xGridRng = xGridMax - xGridMin;
        xGridStep = xGridRng / (nXGrid - 1);
    }

    for (int iX = 0; iX < nXGrid; iX++) {
        xGrid[iX] = xGridMin + iX * xGridStep;
        int iStat=0;
        density_m3[iX] = kj_interp1D(xGrid[iX], &r[0], &n_m3[0], r.size(), iStat);
        if(iStat>0) {
            std::cout << "INTERPOLATION ERROR for Density" << std::endl;
            exit(1);
        }
        iStat=0;
        C3<float> this_b0 = kj_interp1D(xGrid[iX], &r[0], &b0_CYL[0], r.size(), iStat);
        if(iStat>0) {
            std::cout << "INTERPOLATION ERROR for B0" << std::endl;
            exit(1);
        }
        bMag_kjGrid[iX] = mag(this_b0);
        T_keV[iX] = T_keV_cfg; // kj_interp1D(xGrid[iX],r,n_m3);
    }

    float MaxB0 = *max_element(bMag_kjGrid.begin(), bMag_kjGrid.end());

#if USEPAPI >= 1
    cpuTime0 = cpuTime;
    realTime0 = realTime;
    flpIns0 = flpIns;
    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    printf("\nStartup performance:\n");
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif

    float nRFCycles = cfg.lookup("nRFCycles");
    float nStepsPerCyclotronPeriod = cfg.lookup("nStepsPerCyclotronPeriod");
    float tRF = (2 * physConstants::pi) / wrf;
    int nPhi = cfg.lookup("nPhi");
    float ky = cfg.lookup("ky"); // Only used for -DCYLINDRICAL_INPUT_FIELDS=0
    float kz = cfg.lookup("kz"); // Only used for -DCYLINDRICAL_INPUT_FIELDS=0
    int istat = 0;
    int nPx = cfg.lookup("nP_Vx");
    int nPy = cfg.lookup("nP_Vy");
    int nPz = cfg.lookup("nP_Vz");
    float amu = cfg.lookup("species_amu");
    float Z = cfg.lookup("species_Z");
    int nThermal = cfg.lookup("nThermal");
    long int nP = nPx * nPy * nPz;
    float wc = std::abs ( Z * physConstants::e * MaxB0 / (amu * physConstants::mi) );
    float cyclotronPeriod = 2 * physConstants::pi / wc;
    float dtMin = -cyclotronPeriod / nStepsPerCyclotronPeriod;

    int SanityCheck = 0;

    if (std::isinf(cyclotronPeriod)) ++SanityCheck;

    if (SanityCheck > 0) {
        std::cout<<"SanityCheck Failure"<<std::endl;
        exit(SanityCheck);
    }

    int nSteps = nRFCycles * tRF / std::abs(dtMin) + 1;

    for (int iX = 0; iX < nXGrid; iX++) {
        float this_wc = Z * physConstants::e * bMag_kjGrid[iX] / (amu * physConstants::amu);
        wrf_wc[iX] = wrf / this_wc;
        std::cout<<"mass: "<<amu*physConstants::mi<<std::endl;
        std::cout<<"wrf_wc[iX]: "<<wrf_wc[iX]<<std::endl;
    }

#if PRINT_INFO >= 1
    std::cout << "dtMin [s]: " << dtMin << std::endl;
    std::cout << "Cyclotron Period: " << cyclotronPeriod << std::endl;
    std::cout << "RF Period: " << tRF << std::endl;
    std::cout << "nSteps: " << nSteps << std::endl;
    std::cout << "nStepsPerCyclotronPeriod: " << nStepsPerCyclotronPeriod << std::endl;
    std::cout << "freq: " << freq << std::endl;
    std::cout << "Max B0: " << MaxB0 << std::endl;
#endif

    vector<float> thisT;
    try {
        thisT.resize(nSteps);
    } catch (const std::bad_alloc& error) {
        std::cout << "Allocation error at " << __FILE__ << __LINE__ << std::endl;
        std::cout << error.what();
    }

    for (int i = 0; i < nSteps; i++) {
        thisT[i] = i * dtMin; //+1.5*dtMin;
    }

    vector<float> hanningWeight(nSteps);
    vector<float> expWeight(nSteps);
    vector<float> linearWeight(nSteps);
    for (int i = 0; i < nSteps; i++) {
        // linearWeight[i]=thisT[i]*1.0/(tRF*nRFCycles)+1.0;
        hanningWeight[i] = 0.5 * (1 - cos(2 * physConstants::pi * i / (nSteps - 1))); // Regular
        // hanningWeight[i]=0.5*(1-cos(2*physConstants::pi*i/(nSteps*0.25-1))); //Sharper
        // hanningWeight[i] = linearWeight[i];
        if (i < nSteps / 2)
            hanningWeight[i] = 1; // Regular
        // if(i<nSteps*7.0/8.0) hanningWeight[i]=1; //Sharper
        // complex<float> _i (0.0,1.0);
        // complex<float> wrf_c (wrf,wrf*0.0025);
        // expWeight[i] = 1.0;//std::abs(exp(-_i*wrf_c*thisT[i]));
        // hanningWeight[i] = hanningWeight[i] * expWeight[i];
    }

    vector<vector<float> > j1x(nXGrid), j1y(nXGrid), j1z(nXGrid);
    vector<complex<float> > j1xc(nXGrid), j1yc(nXGrid), j1zc(nXGrid);

#if defined(_OPENMP)
    int nThreads, tid, spoken = 0;
#endif

#if CLOCK >= 1
        clock_t startTimeFunctor = clock();
#endif

    float dv;

    // Create worklist of nX * nP particles

    long int nWork = nXGrid * nP;

    vector<CParticle> particleWorkList;
    for (int iX = 0; iX < nXGrid; iX++) {

        vector<CParticle> moreWork(
            create_particles(xGrid[iX], amu, Z, T_keV[iX], density_m3[iX], nPx, nPy,
                nPz, nThermal, dv, &r[0], &b0_CYL[0], r.size() ));

        particleWorkList.insert( particleWorkList.end(), moreWork.begin(), moreWork.end() );
    }

#ifdef __HIPCC__
    std::cout<<"Copying particle worklist to device ..."<<std::endl;
    thrust::device_vector<CParticle> particleWorkList_device = particleWorkList;

    thrust::device_vector<float> vx_device(nWork,0);
    thrust::device_vector<float> vy_device(nWork,0);
    thrust::device_vector<float> vz_device(nWork,0);

    thrust::transform( vx_device.begin(), vx_device.end(), particleWorkList_device.begin(), vx_device.begin(), set_vx() );
    thrust::transform( vy_device.begin(), vy_device.end(), particleWorkList_device.begin(), vy_device.begin(), set_vy() );
    thrust::transform( vz_device.begin(), vz_device.end(), particleWorkList_device.begin(), vz_device.begin(), set_vz() );

    std::cout<<"DONE"<<std::endl;
#endif

    vector<float> vx(nWork,0);
    vector<float> vy(nWork,0);
    vector<float> vz(nWork,0);

    transform( vx.begin(), vx.end(), particleWorkList.begin(), vx.begin(), set_vx() );
    transform( vy.begin(), vy.end(), particleWorkList.begin(), vy.begin(), set_vy() );
    transform( vz.begin(), vz.end(), particleWorkList.begin(), vz.begin(), set_vz() );

    // Velocity space calculation

    vector<C3<float> > df0_dv_XYZ(nWork,0);
    vector<C3<std::complex<float> > > E1(nWork,0);
    vector<C3<std::complex<float> > > B1(nWork,0);
    vector<C3<std::complex<float> > > vCrossB(nWork,0);
    vector<C3<std::complex<float> > > vCrossB_E1(nWork,0);
    vector<complex<float> > forceDotGradf0(nWork,0);
    vector<complex<float> > dtIntegral(nWork,0);
    vector<complex<float> > f1(nWork,0);
    vector<complex<float> > vxf1(nWork,0);
    vector<complex<float> > vyf1(nWork,0);
    vector<complex<float> > vzf1(nWork,0);

#ifdef __HIPCC__

    thrust::device_vector<C3<float> > df0_dv_XYZ_device(nWork,0);
    thrust::device_vector<C3<thrust::complex<float> > > E1_device(nWork,0);
    thrust::device_vector<C3<thrust::complex<float> > > B1_device(nWork,0);
    thrust::device_vector<C3<thrust::complex<float> > > vCrossB_device(nWork,0);
    thrust::device_vector<C3<thrust::complex<float> > > vCrossB_E1_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > forceDotGradf0_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > dtIntegral_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > f1_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > vxf1_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > vyf1_device(nWork,0);
    thrust::device_vector<thrust::complex<float> > vzf1_device(nWork,0);

    thrust::host_vector<CParticle> p_host(nWork);
    thrust::host_vector<C3<float> > df0_dv_XYZ_host(nWork,0);
    thrust::host_vector<C3<thrust::complex<float> > > E1_host(nWork,0);
    thrust::host_vector<C3<thrust::complex<float> > > B1_host(nWork,0);
    thrust::host_vector<C3<thrust::complex<float> > > vCrossB_host(nWork,0);
    thrust::host_vector<C3<thrust::complex<float> > > vCrossB_E1_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > forceDotGradf0_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > dtIntegral_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > f1_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > vxf1_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > vyf1_host(nWork,0);
    thrust::host_vector<thrust::complex<float> > vzf1_host(nWork,0);
 
    // Also copy across the fields to be interpolated to the device
    // For some reason, and only for the thrust::complex<> type, it
    // seems required to go via a thrust::host_vector<> rather than
    // directly to the thrust::device_vector<> as with the floats.

    thrust::host_vector<C3<thrust::complex<float> > > e1_CYL_host(e1_CYL);
    thrust::host_vector<C3<thrust::complex<float> > > b1_CYL_host(b1_CYL);

    thrust::device_vector<float> r_device = r;
    thrust::device_vector<C3<float> > b0_CYL_device = b0_CYL;
    thrust::device_vector<C3<thrust::complex<float> > > e1_CYL_device = e1_CYL_host;
    thrust::device_vector<C3<thrust::complex<float> > > b1_CYL_device = b1_CYL_host;

    float *r_dPtr_raw = thrust::raw_pointer_cast(r_device.data());
    C3<float> *b0_dPtr_raw = thrust::raw_pointer_cast(b0_CYL_device.data());
    C3<thrust::complex<float> > *e1_dPtr_raw = thrust::raw_pointer_cast(e1_CYL_device.data());
    C3<thrust::complex<float> > *b1_dPtr_raw = thrust::raw_pointer_cast(b1_CYL_device.data());

#endif

    // Move particles
    std::cout << "Moving particles with for_each ..." << std::endl;

    for (int i = 0; i < nSteps; i++) {

        //std::cout<<"Step "<<i<<" of "<<nSteps<<std::endl;

        float dtIntFac = 1;
        if (i > 0) dtIntFac = 2;

        dtIntFac = dtMin / 2.0 * dtIntFac;

#ifdef __HIPCC__
        // Move particle
        thrust::for_each( particleWorkList_device.begin(), particleWorkList_device.end(), 
                        moveParticle(dtMin, r_dPtr_raw, b0_dPtr_raw, r.size()) ); 
        thrust::copy(particleWorkList_device.begin(),particleWorkList_device.end(),p_host.begin());

        // df0(v)/dv 
        thrust::transform( particleWorkList_device.begin(), particleWorkList_device.end(), df0_dv_XYZ_device.begin(), 
                        get_df0_dv() ); 
        thrust::copy(df0_dv_XYZ_device.begin(),df0_dv_XYZ_device.end(),df0_dv_XYZ_host.begin());

        // E1(x) 
        thrust::transform( particleWorkList_device.begin(), particleWorkList_device.end(), E1_device.begin(), 
                        getPerturbedField_device(r_dPtr_raw,e1_dPtr_raw,r.size(),nPhi,ky,kz,hanningWeight[i],wrf,thisT[i]) ); 
        thrust::copy(E1_device.begin(),E1_device.end(),E1_host.begin());

        // B1(x) 
        thrust::transform( particleWorkList_device.begin(), particleWorkList_device.end(), B1_device.begin(), 
                        getPerturbedField_device(r_dPtr_raw,b1_dPtr_raw,r.size(),nPhi,ky,kz,hanningWeight[i],wrf,thisT[i]) ); 
        thrust::copy(B1_device.begin(),B1_device.end(),B1_host.begin());

        // v x B1 
        thrust::transform( particleWorkList_device.begin(), particleWorkList_device.end(), B1_device.begin(), vCrossB_device.begin(), 
                        vCross<thrust::complex<float> >() );
        thrust::copy(vCrossB_device.begin(),vCrossB_device.end(),vCrossB_host.begin());

        // E1 + v x B1
        thrust::transform( E1_device.begin(), E1_device.end(), vCrossB_device.begin(), vCrossB_E1_device.begin(), 
                        thrust::plus<C3<thrust::complex<float> > >() );
        thrust::copy(vCrossB_E1_device.begin(),vCrossB_E1_device.end(),vCrossB_E1_host.begin());

        //  (E1 + v x B1) . grad_v(f0(v))
        thrust::transform( vCrossB_E1_device.begin(), vCrossB_E1_device.end(), df0_dv_XYZ_device.begin(), forceDotGradf0_device.begin(), 
                        doDotProduct_device() );
        thrust::copy(forceDotGradf0_device.begin(),forceDotGradf0_device.end(),forceDotGradf0_host.begin());

        // int( (E1 + v x B1) . grad_v(f0(v)), dt ) via running dt integral
        thrust::transform( dtIntegral_device.begin(), dtIntegral_device.end(), forceDotGradf0_device.begin(), dtIntegral_device.begin(), 
                        runningIntegral<thrust::complex<float> >(dtIntFac) );
        thrust::copy(dtIntegral_device.begin(),dtIntegral_device.end(),dtIntegral_host.begin());

        // f1(v) = -q/m * int( (E1 + v x B1) . grad_v(f0(v)), dt )
        thrust::transform( dtIntegral_device.begin(), dtIntegral_device.end(), particleWorkList_device.begin(), f1_device.begin(), 
                        multiplyByChargeOverMass<thrust::complex<float> >() ); 
        thrust::copy(f1_device.begin(),f1_device.end(),f1_host.begin());

        // q . f1(v) // first step in velocity momemnt for current calculation 
        thrust::transform( f1_device.begin(), f1_device.end(), particleWorkList_device.begin(), f1_device.begin(), 
                        multiplyByCharge<thrust::complex<float> >() ); 
        // q . vx . f1(v) 
        thrust::transform( f1_device.begin(), f1_device.end(), vx_device.begin(), vxf1_device.begin(), 
                        thrust::multiplies<thrust::complex<float> >() ); 

        thrust::copy(vxf1_device.begin(),vxf1_device.end(),vxf1_host.begin());

        // q . vy . f1(v) 
        thrust::transform( f1_device.begin(), f1_device.end(), vy_device.begin(), vyf1_device.begin(), 
                        thrust::multiplies<thrust::complex<float> >() ); 
        // q . vz . f1(v) 
        thrust::transform( f1_device.begin(), f1_device.end(), vz_device.begin(), vzf1_device.begin(), 
                        thrust::multiplies<thrust::complex<float> >() ); 


#endif 

#if DO_CPU_ITERATOR_APPROACH > 0

        // Move particle
#if GC_ORBITS >= 1
        for_each( particleWorkList.begin(), particleWorkList.end(), 
                        moveParticle_gc(dtMin, thisT[i], &r[0], &b0_CYL[0], r.size(), &r_gc[0], &curv_CYL[0], &grad_CYL[0], &bDotGradB[0], r_gc.size() ) ); 
#else
        for_each( particleWorkList.begin(), particleWorkList.end(), 
                        moveParticle(dtMin, &r[0], &b0_CYL[0], r.size() ) ); 
#endif
#ifdef __HIPCC__
        std::cout<<"move CPU: "<<particleWorkList[0].c1<<particleWorkList[0].c2<<particleWorkList[0].c3
                <<" GPU: "<<p_host[0].c1<<p_host[0].c2<<p_host[0].c3<<std::endl;
#endif
        // df0(v)/dv 
        transform( particleWorkList.begin(), particleWorkList.end(), df0_dv_XYZ.begin(), 
                        get_df0_dv() ); 

#ifdef __HIPCC__
       std::cout<<"df0_dv_XYZ CPU: "<<df0_dv_XYZ[0]<<" GPU: "<<df0_dv_XYZ_host[0]<<std::endl;
#endif
        // E1(x) 
        transform( particleWorkList.begin(), particleWorkList.end(), E1.begin(), 
                        getPerturbedField(&r[0],&e1_CYL[0],r.size(),nPhi,ky,kz,hanningWeight[i],wrf,thisT[i]) ); 
        
#ifdef __HIPCC__
        std::cout<<"E1 CPU: "<<E1[0]<<" GPU: "<<E1_host[0]<<std::endl;
#endif
        // B1(x) 
        transform( particleWorkList.begin(), particleWorkList.end(), B1.begin(), 
                        getPerturbedField(&r[0],&b1_CYL[0],r.size(),nPhi,ky,kz,hanningWeight[i],wrf,thisT[i]) ); 

#ifdef __HIPCC__
        std::cout<<"B1 CPU: "<<B1[0]<<" GPU: "<<B1_host[0]<<std::endl;
#endif
        // v x B1 
        transform( particleWorkList.begin(), particleWorkList.end(), B1.begin(), vCrossB.begin(), 
                        vCross<std::complex<float> >() );

#ifdef __HIPCC__
        std::cout<<"vCross CPU: "<<vCrossB[0]<<" GPU: "<<vCrossB_host[0]<<std::endl;
#endif
        // E1 + v x B1
        transform( E1.begin(), E1.end(), vCrossB.begin(), vCrossB_E1.begin(), 
                        std::plus<C3<std::complex<float> > >() );

#ifdef __HIPCC__
        std::cout<<"vCrosB_E1 CPU: "<<vCrossB_E1[0]<<" GPU: "<<vCrossB_E1_host[0]<<std::endl;
#endif
        //  (E1 + v x B1) . grad_v(f0(v))
        transform( vCrossB_E1.begin(), vCrossB_E1.end(), df0_dv_XYZ.begin(), forceDotGradf0.begin(), 
                        doDotProduct() );

#ifdef __HIPCC__
        std::cout<<"forceDotGradf0 CPU: "<<forceDotGradf0[0]<<" GPU: "<<forceDotGradf0_host[0]<<std::endl;
#endif
        // int( (E1 + v x B1) . grad_v(f0(v)), dt ) via running dt integral
        transform( dtIntegral.begin(), dtIntegral.end(), forceDotGradf0.begin(), dtIntegral.begin(), 
                        runningIntegral<std::complex<float> >(dtIntFac) );
#ifdef __HIPCC__
        std::cout<<"dtIntegral CPU: "<<dtIntegral[0]<<" GPU: "<<dtIntegral_host[0]<<std::endl;
#endif
        // f1(v) = -q/m * int( (E1 + v x B1) . grad_v(f0(v)), dt )
        transform( dtIntegral.begin(), dtIntegral.end(), particleWorkList.begin(), f1.begin(), 
                        multiplyByChargeOverMass<std::complex<float> >() ); 

        // q . f1(v) // first step in velocity momemnt for current calculation 
        transform( f1.begin(), f1.end(), particleWorkList.begin(), f1.begin(), 
                        multiplyByCharge<std::complex<float> >() ); 

        // q . vx . f1(v) 
        transform( f1.begin(), f1.end(), vx.begin(), vxf1.begin(), 
                        std::multiplies< complex<float> >() ); 

#ifdef __HIPCC__
        std::cout<<"CPU: "<<vxf1[0]<<" GPU: "<<vxf1_host[0]<<std::endl;
#endif
        // q . vy . f1(v) 
        transform( f1.begin(), f1.end(), vy.begin(), vyf1.begin(), 
                        std::multiplies< complex<float> >() ); 

        // q . vz . f1(v) 
        transform( f1.begin(), f1.end(), vz.begin(), vzf1.begin(), 
                        std::multiplies< complex<float> >() ); 
#endif

    }

    // Reduce velocity space to current via the first velocity moment

#if DO_CPU_ITERATOR_APPROACH > 0
    for (int i=0;i<nXGrid;i++) {
        j1xc[i] = dv * accumulate( vxf1.begin()+nP*i, vxf1.begin()+nP*i+nP, complex<float>(0) );
        j1yc[i] = dv * accumulate( vyf1.begin()+nP*i, vyf1.begin()+nP*i+nP, complex<float>(0) );
        j1zc[i] = dv * accumulate( vzf1.begin()+nP*i, vzf1.begin()+nP*i+nP, complex<float>(0) );
        std::cout << j1xc[i].real() << "  " << j1xc[i].imag() << std::endl;
    }
#endif

#ifdef __HIPCC__

    // Copy data back from GPU

    thrust::copy(vxf1_device.begin(),vxf1_device.end(),vxf1_host.begin());
    thrust::copy(vyf1_device.begin(),vyf1_device.end(),vyf1_host.begin());
    thrust::copy(vzf1_device.begin(),vzf1_device.end(),vzf1_host.begin());

    for (int i=0;i<nXGrid;i++) {
        j1xc[i] = dv * accumulate( vxf1_host.begin()+nP*i, vxf1_host.begin()+nP*i+nP, thrust::complex<float>(0) );
        j1yc[i] = dv * accumulate( vyf1_host.begin()+nP*i, vyf1_host.begin()+nP*i+nP, thrust::complex<float>(0) );
        j1zc[i] = dv * accumulate( vzf1_host.begin()+nP*i, vzf1_host.begin()+nP*i+nP, thrust::complex<float>(0) );
        std::cout << j1xc[i].real() << "  " << j1xc[i].imag() << std::endl;
    }
#endif

    stringstream ncjPFileName2("output/jP2.nc");

    NcFile ncjPFile(ncjPFileName2.str().c_str(), NcFile::replace);

    NcDim nc_nX = ncjPFile.addDim("nJp", nXGrid);

    NcVar nc_x = ncjPFile.addVar("x", ncFloat, nc_nX);

    NcVar nc_j1xc_re = ncjPFile.addVar("j1xc_re", ncFloat, nc_nX);
    NcVar nc_j1xc_im = ncjPFile.addVar("j1xc_im", ncFloat, nc_nX);

    NcVar nc_j1yc_re = ncjPFile.addVar("j1yc_re", ncFloat, nc_nX);
    NcVar nc_j1yc_im = ncjPFile.addVar("j1yc_im", ncFloat, nc_nX);

    NcVar nc_j1zc_re = ncjPFile.addVar("j1zc_re", ncFloat, nc_nX);
    NcVar nc_j1zc_im = ncjPFile.addVar("j1zc_im", ncFloat, nc_nX);

    vector<float> JxRe(nXGrid,0);
    vector<float> JxIm(nXGrid,0);
    vector<float> JyRe(nXGrid,0);
    vector<float> JyIm(nXGrid,0);
    vector<float> JzRe(nXGrid,0);
    vector<float> JzIm(nXGrid,0);

    for (int i=0;i<nXGrid;i++) {
       JxRe[i] = j1xc[i].real(); 
       JxIm[i] = j1xc[i].imag(); 
       JyRe[i] = j1yc[i].real(); 
       JyIm[i] = j1yc[i].imag(); 
       JzRe[i] = j1zc[i].real(); 
       JzIm[i] = j1zc[i].imag(); 
    }
    nc_x.putVar(&xGrid[0]);
    nc_j1xc_re.putVar(&JxRe[0]);
    nc_j1xc_im.putVar(&JxIm[0]);
    nc_j1yc_re.putVar(&JyRe[0]);
    nc_j1yc_im.putVar(&JyIm[0]);
    nc_j1zc_re.putVar(&JzRe[0]);
    nc_j1zc_im.putVar(&JzIm[0]);

    std::cout << "DONE" << std::endl;

#if DO_CPU_APPROACH > 0

#if CLOCK >= 1
#if not defined(_OPENMP)
        clock_t endTimeFunctor = clock();
        double timeInSecondsFunctor = (endTimeFunctor - startTimeFunctor) / (double)CLOCKS_PER_SEC;
        std::cout << "Time for this spatial point: " << timeInSecondsFunctor << std::endl;
        std::cout << "Time per particle: " << timeInSecondsFunctor / nWork << std::endl;
#endif
#endif

std::cout << "Continuing with non functor approach ..." << std::endl;

int write_iX = 0;//31;//15;
int write_iP = 180;//52;//33;

#pragma omp parallel for private(istat, tid, spoken)
    for (int iX = 0; iX < nXGrid; iX++) {

#if defined(_OPENMP)
        nThreads = omp_get_num_threads();
        tid = omp_get_thread_num();
        if (tid == 0 && spoken == 0) {
            std::cout << "tid : " << tid << std::endl;
            std::cout << "OMP_NUM_THREADS: " << nThreads << std::endl;
            spoken = 1;
        }
#endif
        vector<CParticle> ThisParticleList(
            create_particles(xGrid[iX], amu, Z, T_keV[iX], density_m3[iX], nPx, nPy,
                nPz, nThermal, dv, &r[0], &b0_CYL[0], r.size() ));

#if CLOCK >= 1
        clock_t startTime = clock();
#endif
        j1xc[iX] = complex<float>(0, 0);
        j1yc[iX] = complex<float>(0, 0);
        j1zc[iX] = complex<float>(0, 0);

#if LOWMEM_USEPAPI >= 1
        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
#endif

#if F1_WRITE >= 1
        int f1_write_iX = write_iX;
        ofstream f1File;
        if (iX == f1_write_iX) {
            f1File.open("output/f1.txt", ios::out | ios::trunc);
            f1File << " vx  vy  vz  re(f1) im(f1) " << std::endl;
        }
#endif

        vector<float> f1(nP);
        vector<complex<float> > f1c(nP);

        for (int iP = 0; iP < nP; iP++) {

            vector<C3<float> > thisOrbitE1_re_XYZ(nSteps, C3<float>(0, 0, 0));
            vector<C3<float> > thisOrbitE1_im_XYZ(nSteps, C3<float>(0, 0, 0));

            vector<C3<float> > thisOrbitB1_re_XYZ(nSteps, C3<float>(0, 0, 0));
            vector<C3<float> > thisOrbitB1_im_XYZ(nSteps, C3<float>(0, 0, 0));

            CParticle thisParticle_XYZ(ThisParticleList[iP]);

            float qOverm = thisParticle_XYZ.q / thisParticle_XYZ.m;

            float Ze = thisParticle_XYZ.q;
#if LOWMEM_ORBIT_WRITE >= 1
            ofstream OrbitFile;
            ofstream v1File;
            ofstream e1_dot_grad_File;
            ofstream df0dv_File;

            if (iX == write_iX && iP == write_iP) {
                std::cout << "Write Particle Properties:" << std::endl;
                std::cout << " vTh: " << thisParticle_XYZ.vTh << std::endl;
                std::cout << " v1: " << thisParticle_XYZ.v_c1 << std::endl;
                std::cout << " v2: " << thisParticle_XYZ.v_c2 << std::endl;
                std::cout << " v3: " << thisParticle_XYZ.v_c3 << std::endl;

                OrbitFile.open("output/orbit.txt", ios::out | ios::trunc);
                OrbitFile << "wc / wrf: " << wrf_wc[iX] << std::endl;
                OrbitFile << " t  x  y  z  re(e1)  im(e1)  re(e2)  im(e2)  re(e3)  "
                             "im(e3)  re(b1)  im(b1)  re(b2)  im(b2)  re(b3)  im(b3) "
                             "status"
                          << std::endl;
                v1File.open("output/orbit_v1.txt", ios::out | ios::trunc);
                v1File << " t  re(v11)  im(v11)  re(v12)  im(v12)  re(v13)  im(v13)"
                       << std::endl;
                e1_dot_grad_File.open("output/orbit_e1_dot_grad_df0_dv.txt",
                    ios::out | ios::trunc);
                e1_dot_grad_File << " t  re(e1.Gradvf0)  im(e1.Gradvf0)  re(e1_per.Gradvf0_per)  "
                                    "im(e1_per.Gradvf0_per)  re(e1_par.Gradvf0_par)  im(e1_par.Gradvf0_par)"
                                 << std::endl;
                df0dv_File.open("output/df0dv.txt", ios::out | ios::trunc);
                df0dv_File << " t  vx  vy  vz  valp  vbet  vpar  vper  gyroAngle  "
                              "df0dv_x  df0dv_y  df0dv_z"
                           << std::endl;
            }
#endif
            // generate orbit and get time-harmonic e along it

            vector<C3<float> > thisOrbit_XYZ(nSteps);
            vector<C3<std::complex<float> > > thisE1c_XYZ(nSteps, C3<std::complex<float> >());
            vector<C3<std::complex<float> > > thisB1c_XYZ(nSteps, C3<std::complex<float> >());
            C3<std::complex<float> > thisV1c_(0, 0, 0), dVc(0, 0, 0), crossTerm(0, 0, 0);
            vector<complex<float> > this_e1_dot_gradvf0(nSteps);
            vector<complex<float> > this_e1_dot_gradvf0_parOnly(nSteps);
            vector<complex<float> > this_e1_dot_gradvf0_perOnly(nSteps);
            vector<C3<std::complex<float> > > this_vCrossB1(nSteps);

            for (int i = 0; i < nSteps; i++) {
#if DEBUG_MOVE >= 1
                std::cout << "Position Before Move: " << thisParticle_XYZ.c1 << "  "
                     << thisParticle_XYZ.c2 << "  " << thisParticle_XYZ.c3 << std::endl;
                std::cout << "p.status: " << thisParticle_XYZ.status << std::endl;
#endif
                thisOrbit_XYZ[i] = C3<float>(thisParticle_XYZ.c1, thisParticle_XYZ.c2,
                    thisParticle_XYZ.c3);
#if GC_ORBITS >= 1
                int MoveStatus = rk4_move_gc(thisParticle_XYZ, dtMin, thisT[i], &r[0], &b0_CYL[0], r.size(), 
                                &r_gc[0], &curv_CYL[0], &grad_CYL[0], &bDotGradB[0], r_gc.size());
#else
                int MoveStatus = rk4_move(thisParticle_XYZ, dtMin, &r[0], &b0_CYL[0], r.size());
#endif
                int OverallStatus = max(thisParticle_XYZ.status, MoveStatus);
#if DEBUG_MOVE >= 1
                std::cout << "Position After Move: " << thisParticle_XYZ.c1 << "  "
                         << thisParticle_XYZ.c2 << "  " << thisParticle_XYZ.c3 << std::endl;
                if (MoveStatus > 0) {
                    std::cout << "MoveStatus: " << MoveStatus << std::endl;
                }
#endif
                C3<float> thisPos_XYZ(thisParticle_XYZ.c1, thisParticle_XYZ.c2, thisParticle_XYZ.c3);
                C3<float> thisPos_CYL = XYZ_to_CYL(thisPos_XYZ);
                C3<float> thisB0 = kj_interp1D(thisPos_CYL.c1, &r[0], &b0_CYL[0], r.size(), thisParticle_XYZ.status);

#if GC_ORBITS >= 1
                C3<float> par = thisB0 / mag(thisB0);
                C3<float> per = cross(par,C3<float>(1,0,0));
                per = per / mag(per);

                // We can just pick any gyrophase and use that to compute gradv_f0, 
                // since gradvf0.par is independent of gyrophase (proven in the commented
                // section below)

                C3<float> this_v = par * thisParticle_XYZ.vPar + per * thisParticle_XYZ.vPer;
                C3<float> this_gradv_f0_GC = ( maxwellian_df0_dv(this_v, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z) );
                float this_gradv_f0_GC_dot_par = dot(this_gradv_f0_GC,par);
 
                //// Prove that gradv_f0_par is independent of gyrophase
                //std::vector< C3<float> > gradv_f0_XYZ_GC;
                //int nTh = 12;
                //float dTh = 360.0 / nTh;
                //for(int iTh=0; iTh<nTh; iTh++){
                //        float th = iTh*dTh;
                //        C3<float> thisPer = rot_axis_angle(per,par,th); 
                //        C3<float> this_v_2 = par * thisParticle_XYZ.vPar + thisPer * thisParticle_XYZ.vPer;

                //        gradv_f0_XYZ_GC.push_back( maxwellian_df0_dv(this_v_2, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z) );
                //        if(thisParticle_XYZ.status>0) gradv_f0_XYZ_GC[iTh] = 0; // To account for the / 0 above.

                //        std::cout<<"iTh: "<<iTh<<std::endl;
                //        std::cout<<"th: "<<th<<std::endl;
                //        std::cout<<"thisPer: "<<thisPer<<std::endl;
                //        std::cout<<"mag(thisPer): "<<mag(thisPer)<<std::endl;
                //        std::cout<<"mag(thisVel): "<<mag(this_v_2)<<std::endl;
                //        std::cout<<"mag(v): "<<std::sqrt(std::pow(thisParticle_XYZ.vPar,2)+std::pow(thisParticle_XYZ.vPer,2))<<std::endl;
                //        std::cout<<"this_v_2: "<<this_v_2<<std::endl;
                //        std::cout<<"grad_f0: "<<gradv_f0_XYZ_GC[iTh]<<std::endl;
                //        std::cout<<"grad_f0: "<<maxwellian_df0_dv(this_v_2, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z)<<std::endl;
                //        std::cout<<"grad_f0.par: "<<dot(gradv_f0_XYZ_GC[iTh],par)<<std::endl;

                //        C3<float> this_v_2_above = par * thisParticle_XYZ.vPar + per * thisParticle_XYZ.vPer;
                //        C3<float> this_gradv_f0_GC_above = ( maxwellian_df0_dv(this_v_2_above, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z) );
                //        float this_gradv_f0_GC_dot_par = dot(this_gradv_f0_GC_above,par);
                //        std::cout<<"grad_f0.par (actual): "<<this_gradv_f0_GC_dot_par<<std::endl;

                //}

                //exit(1);

#if DEBUG_MOVE >= 2
                std::cout << "vPar: " << thisParticle_XYZ.vPar << " vPer: " << thisParticle_XYZ.vPer << std::endl;
                std::cout << "status: " << thisParticle_XYZ.status << std::endl;
                std::cout << "c1: " << thisOrbit_XYZ[i].c1 << std::endl;
                std::cout << "thisB0: " << thisB0 << std::endl;
                std::cout << "mag(thisB0): " << mag(thisB0) << std::endl;
                std::cout << "thisParticle_XYZ.vPar: " << thisParticle_XYZ.vPar << std::endl;
                std::cout << "thisVel_XYZ: " << thisVel_XYZ << std::endl;
#endif

#else
                C3<float> thisVel_XYZ(thisParticle_XYZ.v_c1, thisParticle_XYZ.v_c2, thisParticle_XYZ.v_c3);
                C3<float> gradv_f0_XYZ = maxwellian_df0_dv(thisVel_XYZ, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z);
                if(thisParticle_XYZ.status>0) gradv_f0_XYZ = 0; // To account for the / 0 above.
#endif

                complex<float> _i(0, 1);

                // why is this exp(-iwt) here? surely it's not required for the freq domain calc?

                C3<std::complex<float> > E1_XYZ;
#if CYLINDRICAL_INPUT_FIELDS >=1 
                E1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ_fromCYL(thisParticle_XYZ, &r[0], &e1_CYL[0], r.size(), nPhi);
#else
                E1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ_fromXYZ(thisParticle_XYZ, &r[0], &e1_CYL[0], r.size(), ky, kz);
#endif
                thisE1c_XYZ[i] = E1_XYZ * (1 - thisParticle_XYZ.status);

                C3<std::complex<float> > B1_XYZ;
#if CYLINDRICAL_INPUT_FIELDS >=1 
                B1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ_fromCYL(thisParticle_XYZ, &r[0], &b1_CYL[0], r.size(), nPhi);
#else
                B1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ_fromXYZ(thisParticle_XYZ, &r[0], &b1_CYL[0], r.size(), ky, kz);
#endif
                thisB1c_XYZ[i] = B1_XYZ * (1 - thisParticle_XYZ.status);

                //if (iX == write_iX && iP == write_iP) {
                //    std::cout<<"E1_XYZ: "<<E1_XYZ<<std::endl;
                //    std::cout<<"status: "<<thisParticle_XYZ.status<<std::endl;
                //    std::cout<<"hanningWeight: "<<hanningWeight[i]<<std::endl;
                //}

#if DEBUG_MOVE >= 2
                std::cout << "thisE1c[i].c1: " << thisE1c_XYZ[i].c1 << std::endl;
                std::cout << "thisE1c[i].c2: " << thisE1c_XYZ[i].c2 << std::endl;
                std::cout << "thisE1c[i].c3: " << thisE1c_XYZ[i].c3 << std::endl;

                std::cout << "thisB1c[i].c1: " << thisB1c_XYZ[i].c1 << std::endl;
                std::cout << "thisB1c[i].c2: " << thisB1c_XYZ[i].c2 << std::endl;
                std::cout << "thisB1c[i].c3: " << thisB1c_XYZ[i].c3 << std::endl;
#endif
#if DEBUG_FORCE_TERM >= 1
                std::cout << "thisE1c[i].c1: " << thisE1c_XYZ[i].c1 << std::endl;
                std::cout << "thisE1c[i].c2: " << thisE1c_XYZ[i].c2 << std::endl;
                std::cout << "thisE1c[i].c3: " << thisE1c_XYZ[i].c3 << std::endl;

                std::cout << "thisB1c[i].c1: " << thisB1c_XYZ[i].c1 << std::endl;
                std::cout << "thisB1c[i].c2: " << thisB1c_XYZ[i].c2 << std::endl;
                std::cout << "thisB1c[i].c3: " << thisB1c_XYZ[i].c3 << std::endl;

                std::cout << "thisVel_XYZ.c1: " << thisVel_XYZ.c1 << std::endl;
                std::cout << "thisVel_XYZ.c2: " << thisVel_XYZ.c2 << std::endl;
                std::cout << "thisVel_XYZ.c3: " << thisVel_XYZ.c3 << std::endl;

#endif

#if GC_ORBITS >= 1

                C3<std::complex<float> > this_force = thisE1c_XYZ[i];
                this_e1_dot_gradvf0[i] = dot(this_force,par) * this_gradv_f0_GC_dot_par;
                if(thisParticle_XYZ.status>0) this_e1_dot_gradvf0[i] = 0; // To account for the / 0 above.

#else
                this_vCrossB1[i] = cross(thisVel_XYZ, thisB1c_XYZ[i]);
                ////C3<std::complex<float> > this_force = this_vCrossB1[i] + thisE1c_XYZ[i];
                C3<std::complex<float> > this_force = thisE1c_XYZ[i];
                this_e1_dot_gradvf0[i] = dot(this_force, gradv_f0_XYZ);
                if(thisParticle_XYZ.status>0) this_e1_dot_gradvf0[i] = 0; // To account for the / 0 above.

                // Get the per and par contributions to the total e1DotGradvF0 for debugging 

                C3<float> par = thisB0 / mag(thisB0);
                C3<float> per = cross(par,C3<float>(1,0,0));
                per = per / mag(per);

                C3<std::complex<float> > par_force = par*dot(this_force,par);
                C3<float> par_gradf = par*dot(gradv_f0_XYZ,par);

                C3<std::complex<float> > per_force = this_force - par_force;
                C3<float> per_gradf = gradv_f0_XYZ - par_gradf;

                this_e1_dot_gradvf0_perOnly[i] = dot(per_force, per_gradf);
                this_e1_dot_gradvf0_parOnly[i] = dot(par_force, par_gradf);

#endif

#if LOWMEM_ORBIT_WRITE >= 1
#if GC_ORBITS == 0
                if (iX == write_iX && iP == write_iP) {
                    df0dv_File << scientific;
                    df0dv_File << thisT[i] << "    " << thisVel_XYZ.c1 << "    "
                               << thisVel_XYZ.c2 << "    " << thisVel_XYZ.c3 << "    "
                               << thisParticle_XYZ.vAlp << "    " << thisParticle_XYZ.vBet
                               << "    " << thisParticle_XYZ.vPar << "    "
                               << thisParticle_XYZ.vPer << "    " << thisParticle_XYZ.phs
                               << "    " << gradv_f0_XYZ.c1 << "    " << gradv_f0_XYZ.c2
                               << "    " << gradv_f0_XYZ.c3 << std::endl;
                }
#endif
                if (iX == write_iX && iP == write_iP) {
                    OrbitFile << scientific;
                    OrbitFile << thisT[i] << "    " 
                              << thisPos_XYZ.c1 << "    " 
                              << thisPos_XYZ.c2 << "    " 
                              << thisPos_XYZ.c3 << "    " 
                              << real(thisE1c_XYZ[i].c1) << "    " 
                              << imag(thisE1c_XYZ[i].c1) << "    "
                              << real(thisE1c_XYZ[i].c2) << "    "
                              << imag(thisE1c_XYZ[i].c2) << "    "
                              << real(thisE1c_XYZ[i].c3) << "    "
                              << imag(thisE1c_XYZ[i].c3) << "    "
                              << real(thisB1c_XYZ[i].c1) << "    "
                              << imag(thisB1c_XYZ[i].c1) << "    "
                              << real(thisB1c_XYZ[i].c2) << "    "
                              << imag(thisB1c_XYZ[i].c2) << "    "
                              << real(thisB1c_XYZ[i].c3) << "    "
                              << imag(thisB1c_XYZ[i].c3) << "    "
                              << real(this_vCrossB1[i].c1) << "    "
                              << imag(this_vCrossB1[i].c1) << "    "
                              << real(this_vCrossB1[i].c2) << "    "
                              << imag(this_vCrossB1[i].c2) << "    "
                              << real(this_vCrossB1[i].c3) << "    "
                              << imag(this_vCrossB1[i].c3) << "    "
                              << thisParticle_XYZ.status << std::endl;
                }
                if (iX == write_iX && iP == write_iP) {
                    e1_dot_grad_File << scientific;
                    e1_dot_grad_File << thisT[i] 
                                    << "    " << real(this_e1_dot_gradvf0[i])
                                    << "    " << imag(this_e1_dot_gradvf0[i]) 
                                    << "    " << real(this_e1_dot_gradvf0_perOnly[i])
                                    << "    " << imag(this_e1_dot_gradvf0_perOnly[i])
                                    << "    " << real(this_e1_dot_gradvf0_parOnly[i])
                                    << "    " << imag(this_e1_dot_gradvf0_parOnly[i])
                                    << std::endl;
                }
#endif
            }
#if LOWMEM_ORBIT_WRITE >= 1
            if (iX == write_iX && iP == write_iP) {
                OrbitFile.close();
            }
#endif

            complex<float> this_f1c = -qOverm * intVecArray(thisT, this_e1_dot_gradvf0);

#if GC_ORBITS >= 1

            // Add the offset to the GC time integration 

            C3<float> StartingPos_XYZ(ThisParticleList[iP].c1, ThisParticleList[iP].c2, ThisParticleList[iP].c3);
            C3<float> StartingPos_CYL = XYZ_to_CYL(StartingPos_XYZ)
 
            C3<float> thisB0 = kj_interp1D(StartingPos_CYL.c1, &r[0], &b0_CYL[0], r.size(), thisParticle_XYZ.status);
            C3<float> par = thisB0 / mag(thisB0);
            C3<float> per = cross(par,C3<float>(1,0,0));
            per = per / mag(per);

            // Calculate the initial gyrophase offset to be added to
            // the guiding center calculation ...

            C3<std::complex<float> > initial_force = thisE1c_XYZ[0];
            C3<float> initialV_XYZ(ThisParticleList[iP].v_c1, ThisParticleList[iP].v_c2, ThisParticleList[iP].v_c3);
            C3<float> initial_gradv_f0_XYZ = maxwellian_df0_dv(initialV_XYZ, T_keV[iX], density_m3[iX], thisParticle_XYZ.amu, thisParticle_XYZ.Z);

            // Angle between perp only components

            C3<std::complex<float> > perp_force = initial_force - par*dot(initial_force,par);
            C3<float> perp_gradf = initial_gradv_f0_XYZ - par*dot(initial_gradv_f0_XYZ,par);

            C3<float> perp_force_re(perp_force.c1.real(),perp_force.c2.real(),perp_force.c3.real());
            C3<float> perp_force_im(perp_force.c1.imag(),perp_force.c2.imag(),perp_force.c3.imag());

            float this_angle_perp_re = std::acos(dot(perp_force_re,perp_gradf) / (mag(perp_force_re)*mag(perp_gradf)));
            float this_angle_perp_im = std::acos(dot(perp_force_im,perp_gradf) / (mag(perp_force_im)*mag(perp_gradf)));

            // what is the total angle, not just the perp one?

            std::complex<float> this_angle_perp = std::complex<float>(this_angle_perp_re,this_angle_perp_im);

            // Normalize angle to the 0<th<360 range

            if(this_angle_perp_re<0) this_angle_perp_re += 2 * physConstants::pi;
            if(this_angle_perp_im<0) this_angle_perp_im += 2 * physConstants::pi;

            // Double check the angle
            if( (this_angle_perp_re<0) || (this_angle_perp_re>2*physConstants::pi) ) {
                    std::cout<<"ERROR: angle out of range"<<std::endl;
                    exit(1);
            }

            if( (this_angle_perp_im<0) || (this_angle_perp_im>2*physConstants::pi) ) {
                    std::cout<<"ERROR: angle out of range"<<std::endl;
                    exit(1);
            }

            float angleEnd_re = 0;
            float angleEnd_im = 0;

            if(this_angle_perp_re>=0) angleEnd_re = physConstants::pi;
            if(this_angle_perp_re>=physConstants::pi) angleEnd_re = 2*physConstants::pi;

            if(this_angle_perp_im>=0) angleEnd_im = physConstants::pi/2;
            if(this_angle_perp_im>=physConstants::pi/2) angleEnd_im = physConstants::pi/2+physConstants::pi;
            if(this_angle_perp_im>=physConstants::pi/2+physConstants::pi) angleEnd_im = physConstants::pi/2+2*physConstants::pi;

            float offsetReal = +(( std::sin(angleEnd_re) - std::sin(this_angle_perp_re) ) * mag(perp_force)*mag(perp_gradf)).real();
            float offsetImag = -(( std::cos(this_angle_perp_im) - std::cos(angleEnd_im) ) * mag(perp_force)*mag(perp_gradf)).imag();

            // Convert gyro angle integral to time integral

            float offset_wc = std::abs(ThisParticleList[iP].q*mag(thisB0)/ThisParticleList[iP].m);
            float offset_period = 2.0f*physConstants::pi/offset_wc;
            std::complex<float> offset_dt = this_angle_perp / float(2.0f*physConstants::pi)*offset_period;
 
            std::complex<float> offset = std::complex<float>(offsetReal,offsetImag) * dtMin * float(2*physConstants::pi);

            // Account for this_force==0 due to hanningWeight==0 at last point, or e1.gradvf0==0.
            if(isnan(this_angle_perp.real())) {
                offset = 0;
            }

            this_f1c += -qOverm * offset;

            if (iX == write_iX && iP == write_iP) {
                    std::cout<<"Offset: "<<offset<<std::endl;
            } 

            //average_e1_dot_gradvf0 = dot(this_force,par) * dot(initial_gradv_f0_XYZ,par);

            if (iX == write_iX && iP == write_iP) {
                std::cout<<"Offset wc: "<<offset_wc<<std::endl;
                std::cout<<"Offset T: "<<offset_period<<std::endl;
                std::cout<<"Offset_dt: "<<offset_dt<<std::endl;
                std::cout<<"dtMin: "<<dtMin<<std::endl;
                std::cout<<"par: "<<par<<std::endl;
                std::cout<<"thisE1c_XYZ: "<<thisE1c_XYZ[0]<<std::endl;
                std::cout<<"this_force: "<<initial_force<<std::endl;
                std::cout<<"this_angle_perp: "<<this_angle_perp*float(180.0f/physConstants::pi)<<std::endl;
                std::cout<<"perp_force: "<<perp_force<<std::endl;
                std::cout<<"perp_gradv: "<<perp_gradf<<std::endl;
                std::cout<<"initial_gradv_f0_XYZ:"<<initial_gradv_f0_XYZ<<std::endl;
            }
#endif

            if (iX == write_iX && iP == write_iP) {
                    //for(int i=0; i<nSteps;i++){
                    //    std::cout<<"this_e1_dot_gradvf0[i]: "<<this_e1_dot_gradvf0[i]<<std::endl;
                    //}
                    std::cout<<"this_f1c: "<<this_f1c<<std::endl;
            }

#if LOWMEM_ORBIT_WRITE >= 1
            if (iX == write_iX && iP == write_iP) {

                complex<float> tmp = 0.0;
                for (int i = 0; i < nSteps; i++) {
                    tmp += -qOverm * this_e1_dot_gradvf0[i] * dtMin;
                    v1File << thisT[i] << "    " << real(tmp) << "    " << imag(tmp)
                           << std::endl;
                }
            }
#endif
            f1c[iP] = -this_f1c;

            float v0x_i = ThisParticleList[iP].v_c1;
            float v0y_i = ThisParticleList[iP].v_c2;
            float v0z_i = ThisParticleList[iP].v_c3;

            float h = dv * Ze;

#pragma omp critical // "atomic" does not work for complex numbers
            {
                j1xc[iX] += h * (v0x_i * f1c[iP]);
                j1yc[iX] += h * (v0y_i * f1c[iP]);
                j1zc[iX] += h * (v0z_i * f1c[iP]);
#if DEBUG_MOVE >= 2
                std::cout << "v0x_i: " << v0x_i << std::endl;
                std::cout << "v0y_i: " << v0y_i << std::endl;
                std::cout << "v0z_i: " << v0z_i << std::endl;

                std::cout << "f1c[iP]: " << f1c[iP] << std::endl;
                std::cout << "qOverm: " << qOverm << std::endl;
                std::cout << "dtMin: " << dtMin << std::endl;

                std::cout << "j1xc[iX]: " << j1xc[iX] << std::endl;
                std::cout << "j1yc[iX]: " << j1yc[iX] << std::endl;
                std::cout << "j1zc[iX]: " << j1zc[iX] << std::endl;
                //exit(1);
#endif
            }

            if (iX == write_iX && iP == write_iP) {
                std::cout<<"iX: "<<iX<<std::endl;
                std::cout<<"iP: "<<iP<<std::endl;
                std::cout<<"h: "<<h<<std::endl;
                std::cout<<"dv: "<<dv<<std::endl;
                std::cout<<"f1c[iP]: "<<f1c[iP]<<std::endl;
                std::cout<<"j1xc[iX]: "<<j1xc[iX]<<std::endl;
                std::cout<<"j1yc[iX]: "<<j1yc[iX]<<std::endl;
                std::cout<<"j1zc[iX]: "<<j1zc[iX]<<std::endl;
            }

#if F1_WRITE >= 1
            if (iX == f1_write_iX) {
                f1File << scientific;
                f1File << showpos;
                f1File << v0x_i << "    " << v0y_i << "    " << v0z_i << "    "
                       << real(f1c[iP]) << "    " << imag(f1c[iP]) << std::endl;
            }
#endif
        }

#if CLOCK >= 1
#if not defined(_OPENMP)
        clock_t endTime = clock();
        double timeInSeconds = (endTime - startTime) / (double)CLOCKS_PER_SEC;
        std::cout << "Time for this spatial point: " << timeInSeconds << std::endl;
        std::cout << "Time per particle: " << timeInSeconds / nP << std::endl;
#endif
#endif

#if LOWMEM_USEPAPI >= 1
        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
        printf("\nLOWMEM Oribit calculation performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif

#if USEPAPI >= 1
        printf("\nGet e(t) and integrate performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            eT_realTime, eT_cpuTime, eT_flpIns, eT_mFlops );
        printf("\nGet v(t) and integrate performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            vT_realTime, vT_cpuTime, vT_flpIns, vT_mFlops );

        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
        printf("\nj(t) performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif


    } // End of xGrid loop

    // Write current(s) to file

    // std::cout << "Writing jP to file ... ";

    for (int iX = 0; iX < nXGrid; iX++) {

        stringstream ncjPFileName;
        ncjPFileName << "output/";
        // check directory exists
        struct stat st;
        if (stat(ncjPFileName.str().c_str(), &st) != 1) {
            int mkDirStat = mkdir(ncjPFileName.str().c_str(),
                S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
        }
        ncjPFileName << "/jP_";
        ncjPFileName << setw(3) << setfill('0') << iX;
        ncjPFileName << ".nc";
#if DEBUGLEVEL >= 1
        std::cout << ncjPFileName.str().c_str() << std::endl;
#endif

        NcFile ncjPFile(ncjPFileName.str().c_str(), NcFile::replace);

        NcDim nc_scalar = ncjPFile.addDim("scalar", 1);

        NcVar nc_x = ncjPFile.addVar("x", ncFloat, nc_scalar);
        NcVar nc_freq = ncjPFile.addVar("freq", ncFloat, nc_scalar);

        NcVar nc_j1xc_re = ncjPFile.addVar("j1xc_re", ncFloat, nc_scalar);
        NcVar nc_j1xc_im = ncjPFile.addVar("j1xc_im", ncFloat, nc_scalar);

        NcVar nc_j1yc_re = ncjPFile.addVar("j1yc_re", ncFloat, nc_scalar);
        NcVar nc_j1yc_im = ncjPFile.addVar("j1yc_im", ncFloat, nc_scalar);

        NcVar nc_j1zc_re = ncjPFile.addVar("j1zc_re", ncFloat, nc_scalar);
        NcVar nc_j1zc_im = ncjPFile.addVar("j1zc_im", ncFloat, nc_scalar);

        nc_x.putVar(&xGrid[iX]);
        nc_freq.putVar(&freq);

        vector<size_t> startp(1, 0);

        float tmpJxRe = real(j1xc[iX]);
        float tmpJxIm = imag(j1xc[iX]);
        nc_j1xc_re.putVar(&tmpJxRe);
        nc_j1xc_im.putVar(&tmpJxIm);

        float tmpJyRe = real(j1yc[iX]);
        float tmpJyIm = imag(j1yc[iX]);
        nc_j1yc_re.putVar(&tmpJyRe);
        nc_j1yc_im.putVar(&tmpJyIm);

        float tmpJzRe = real(j1zc[iX]);
        float tmpJzIm = imag(j1zc[iX]);
        nc_j1zc_re.putVar(&tmpJzRe);
        nc_j1zc_im.putVar(&tmpJzIm);

        if (iX == write_iX) {
            std::cout<<"write_iX"<<std::endl;
        }

        std::cout<<"j1xc[iX]: "<<j1xc[iX]<<std::endl;
        std::cout<<"j1yc[iX]: "<<j1yc[iX]<<std::endl;
        std::cout<<"j1zc[iX]: "<<j1zc[iX]<<std::endl;

    }

    // ProfilerStop();

    std::cout << "DONE" << std::endl;

#if CLOCK >= 1
    clock_t ProgramTime_ = clock();
    double ProgramTimeInSeconds = (ProgramTime_ - ProgramTime) / (double)CLOCKS_PER_SEC;
#if defined(_OPENMP)
    ProgramTimeInSeconds = ProgramTimeInSeconds / nThreads;
    std::cout << "nThreads: " << nThreads << std::endl;
#endif
    std::cout << "Total Time [s]: " << ProgramTimeInSeconds << std::endl;
    std::cout << "Total Time [m]: " << ProgramTimeInSeconds / 60.0 << std::endl;
    std::cout << "Total Time [h]: " << ProgramTimeInSeconds / 3600.0 << std::endl;
#endif

#endif
    return EXIT_SUCCESS;
}
