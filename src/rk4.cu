#include "hip/hip_runtime.h"
#include "rk4.hpp"

// Zero-order orbits
HOST DEVICE
C3<float> rk4_evalf(CParticle& p, const float& t,
    const C3<float>& v_XYZ, const C3<float>& x, float *rVec, C3<float> *b0Vec_CYL, int nR)
{

    C3<float> b0_XYZ;
    b0_XYZ = getB_XYZ(p, rVec, b0Vec_CYL, nR);

    C3<float> v_x_b0 = cross(v_XYZ, b0_XYZ);

    return v_x_b0 * (p.q / p.m);
}

// Zero-order orbits
HOST DEVICE
int rk4_move(CParticle& p, const float& dt, float *r, C3<float> *b0, int nR)
{

    float t0 = 0;            

    C3<float> yn0(p.v_c1, p.v_c2, p.v_c3), xn0(p.c1, p.c2, p.c3);
    C3<float> k1, k2, k3, k4, yn1, x1, x2, x3, x4, xn1;

    k1 = rk4_evalf(p, t0 + 0.0 * dt, yn0, xn0, r, b0, nR) * dt;
    x1 = yn0 * dt;

    k2 = rk4_evalf(p, t0 + 0.5 * dt, yn0 + 0.5 * k1, xn0 + 0.5 * x1, r, b0, nR) * dt;
    x2 = (yn0 + 0.5 * k1) * dt;

    k3 = rk4_evalf(p, t0 + 0.5 * dt, yn0 + 0.5 * k2, xn0 + 0.5 * x2, r, b0, nR) * dt;
    x3 = (yn0 + 0.5 * k2) * dt;

    k4 = rk4_evalf(p, t0 + 1.0 * dt, yn0 + 1.0 * k3, xn0 + 1.0 * x3, r, b0, nR) * dt;
    x4 = (yn0 + 1.0 * k3) * dt;

    yn1 = yn0 + 1.0 / 6.0 * (k1 + 2.0 * k2 + 2.0 * k3 + k4) * (1 - p.status); // the * (1-p.status) sets the move to zero for dead particles;
    xn1 = xn0 + 1.0 / 6.0 * (x1 + 2.0 * x2 + 2.0 * x3 + x4) * (1 - p.status);

    p.c1 = xn1.c1;
    p.c2 = xn1.c2;
    p.c3 = xn1.c3;
    p.v_c1 = yn1.c1;
    p.v_c2 = yn1.c2;
    p.v_c3 = yn1.c3;

#if _PARTICLE_BOUNDARY == 1
// Particle absorbing walls
#elif _PARTICLE_BOUNDARY == 2
    // Periodic
    if (p.c1 < r[0]) {
#if DEBUGLEVEL >= 1
        cout << "Particle went left" << endl;
#endif
        p.c1 = r[nR-1] - (r[0] - p.c1);
    }
    if (p.c1 > r[nR-1]) {
#if DEBUGLEVEL >= 1
        cout << "Particle went right" << endl;
#endif
        p.c1 = r[0] + (p.c1 - r[nR-1]);
    }
#elif _PARTICLE_BOUNDARY == 3
    // Particle reflecting walls
    if (p.c1 < r[0]) {
        cout << "Particle hit the left wall" << endl;
        cout << "r.front(): " << r[0] << endl;
        p.c1 = r[0] + (r[0] - p.c1);
        p.v_c1 = -p.v_c1;
    }
    if (p.c1 > r[nR-1]) {
        cout << "Particle hit the right wall" << endl;
        cout << "r[nR-1]: " << r[nR-1] << endl;
        p.c1 = r[nR-1] - (p.c1 - r[nR-1]);
        p.v_c1 = -p.v_c1;
    }
#endif

#if DEBUGLEVEL >= 3
    cout << "\tdt: " << dt << endl;
    cout << "\tr.front(): " << r.front() << endl;
    cout << "\tr.back(): " << r.back() << endl;
    cout << "\tx0_XYZ: " << xn0.c1 << "  " << xn0.c2 << "  " << xn0.c3 << endl;
    cout << "\tv0_XYZ: " << yn0.c1 << "  " << yn0.c2 << "  " << yn0.c3 << endl;
    cout << "\tx1_XYZ: " << xn1.c1 << "  " << xn1.c2 << "  " << xn1.c3 << endl;
    cout << "\tv1_XYZ: " << yn1.c1 << "  " << yn1.c2 << "  " << yn1.c3 << endl;
    cout << "\tE: " << 0.5 * p.m * sqrt(pow(p.v_c1, 2) + pow(p.v_c2, 2) + pow(p.v_c3, 2)) / _e << endl;
#endif
    return p.status;
}

// Guiding center orbit
HOST DEVICE
int rk4_move_gc(CParticle& p, const float dt, const float t0,
    const float *r_b0, const C3<float> *b0_CYL, int nB, const float *r_GC,
    const C3<float> *curv_CYL, const C3<float> *grad_CYL,
    const float *bDotGradB, int nGC)
{

    C3<float> xn0_XYZ(p.c1, p.c2, p.c3);
    C3<float> xn0 = XYZ_to_CYL(xn0_XYZ);

    float This_vPer = eval_vPer(p, xn0, r_b0, b0_CYL, nB);
#if DEBUG_GC >= 2
    cout << "p.vPer: " << p.vPer << endl;
    cout << "p.vPar: " << p.vPar << endl;
    cout << "This_vPer: " << This_vPer << endl;
    if (isnan(p.vPer))
        exit(1);
#endif
    C3<float> This_vGC = eval_vGC(p, xn0, This_vPer, p.vPar + 0, r_b0, b0_CYL, nB, r_GC, curv_CYL, grad_CYL, nGC);
    float k1_vPar = dt * eval_aPar(p, xn0, r_GC, bDotGradB, nGC);
    C3<float> k1_vgc = dt * This_vGC;
#if DEBUG_GC >= 2
    std::cout<<"k1_vgc: "<<k1_vgc<<std::endl;
    std::cout<<"xn0: " <<xn0<<std::endl;
    cout << "Status: " << p.status << endl;
    if (isnan(k1_vgc) || isinf(k1_vgc) || isnan(xn0) || isinf(xn0) || p.status > 0) {
        p.status = 1;
        return p.status;
    }
#endif
    This_vPer = eval_vPer(p, xn0 + k1_vgc / 2.0, r_b0, b0_CYL, nB);
    This_vGC = eval_vGC(p, xn0 + k1_vgc / 2.0, This_vPer, p.vPar + k1_vPar / 2.0, r_b0, b0_CYL, nB, r_GC, curv_CYL, grad_CYL, nGC);
    float k2_vPar = dt * eval_aPar(p, xn0 + k1_vgc / 2.0, r_GC, bDotGradB, nGC);
    C3<float> k2_vgc = dt * This_vGC;
#if DEBUG_GC >= 2
    std::cout<<"k2_vgc: "<<k2_vgc<<std::endl;
    if (isnan(k2_vgc) || isinf(k2_vgc) || isnan(xn0) || isinf(xn0) || p.status > 0) {
        p.status = 1;
        return p.status;
    }
#endif
    This_vPer = eval_vPer(p, xn0 + k2_vgc / 2.0, r_b0, b0_CYL, nB);
    This_vGC = eval_vGC(p, xn0 + k2_vgc / 2.0, This_vPer, p.vPar + k2_vPar / 2.0, r_b0, b0_CYL, nB, r_GC, curv_CYL, grad_CYL, nGC);
    float k3_vPar = dt * eval_aPar(p, xn0 + k2_vgc / 2.0, r_GC, bDotGradB, nGC);
    C3<float> k3_vgc = dt * This_vGC;
#if DEBUG_GC >= 2
    std::cout<<"k3_vgc: "<<k3_vgc<<std::endl;
    if (isnan(k3_vgc) || isinf(k3_vgc) || isnan(xn0) || isinf(xn0) || p.status > 0) {
        p.status = 1;
        return p.status;
    }
#endif
    This_vPer = eval_vPer(p, xn0 + k3_vgc, r_b0, b0_CYL, nB);
    This_vGC = eval_vGC(p, xn0 + k3_vgc, This_vPer, p.vPar + k3_vPar, r_b0, b0_CYL, nB, r_GC, curv_CYL, grad_CYL, nGC);
    float k4_vPar = dt * eval_aPar(p, xn0 + k3_vgc, r_GC, bDotGradB, nGC);
    C3<float> k4_vgc = dt * This_vGC;
#if DEBUG_GC >= 2
    std::cout<<"k4_vgc: "<<k4_vgc<<std::endl;
    if (isnan(k4_vgc) || isinf(k4_vgc) || isnan(xn0) || isinf(xn0) || p.status > 0) {
        p.status = 1;
        return p.status;
    }
#endif
    float vPar1 = p.vPar + (k1_vPar + 2.0 * k2_vPar + 2.0 * k3_vPar + k4_vPar) / 6.0 * (1 - p.status);
    C3<float> xn1 = xn0 + (k1_vgc + 2.0 * k2_vgc + 2.0 * k3_vgc + k4_vgc) / 6.0 * (1 - p.status);

#if DEBUG_GC >= 1
    if (isnan(xn1) || isinf(xn1)) {
        p.status = 1;
        return p.status;
    }
#endif

    // Update particle with moved position and new vPar & vPer

    float vPer1 = eval_vPer(p, xn1, r_b0, b0_CYL, nB);

    p.vPar = vPar1;
    p.vPer = vPer1;

    C3<float> xn1_XYZ = CYL_to_XYZ(xn1);

    p.c1 = xn1_XYZ.c1;
    p.c2 = xn1_XYZ.c2;
    p.c3 = xn1_XYZ.c3;

    // Update the XYZ velocity also

    int status = 0;
    C3<float> this_b0_CYL = kj_interp1D(xn1.c1, r_b0, b0_CYL, nB, status);
    p.status = max(p.status, status);

    C3<float> this_b0_XYZ = rot_CYL_to_XYZ(xn1.c2, this_b0_CYL, 1);

    C3<float> v_abp;

    float this_wc = p.q * mag(this_b0_CYL) / p.m;
    p.phs = this_wc * t0 + p.gyroPhase;
    v_abp.c1 = GetAlpComp(vPer1, p.phs);
    v_abp.c2 = GetBetComp(vPer1, p.phs);
    v_abp.c3 = vPar1;

    p.vAlp = v_abp.c1;
    p.vBet = v_abp.c2;

    C3<float> this_v_XYZ = rot_XYZ_to_abp(v_abp, this_b0_XYZ, -1);

    p.v_c1 = this_v_XYZ.c1;
    p.v_c2 = this_v_XYZ.c2;
    p.v_c3 = this_v_XYZ.c3;

    return p.status;
}

