#include "hip/hip_runtime.h"
#include "gcTerms.hpp"
#include <cmath>

// Parallel acceleration
float eval_aPar(CParticle& p, C3<float> r, const float *r_GC, const float *bDotGradB, int nGC)
{

    int status = 0;
    float This_bDotGradB = kj_interp1D(r.c1, r_GC, bDotGradB, nGC, status);
    p.status = max(p.status, status);
#if DEBUG_EVAL_APAR >= 1
    if (status > 0) {
        cout << "ERROR 1 in eval_aPar" << endl;
        exit(1);
    }
#endif
    float aPar = -p.u / p.m * This_bDotGradB;
#if DEBUG_EVAL_APRA >= 1
    if (isnan(aPar) || isinf(aPar)) {
        status = 1;
        cout << "ERROR 2 in eval_aPar" << endl;
        exit(1);
    }
#endif
    return aPar;
}

// Perpendicular velocity
float eval_vPer(CParticle& p, C3<float> r, const float *r_b0, const C3<float> *b0_CYL, int n)
{
    int status = 0;
    C3<float> This_b0_CYL = kj_interp1D(r.c1, r_b0, b0_CYL, n, status);
    p.status = max(p.status, status);
    return std::sqrt(2.0 * p.u * mag(This_b0_CYL) / p.m);
}

// Guiding center veclocity
C3<float> eval_vGC(CParticle& p, C3<float> r, const float vPer, const float vPar,
    const float *r_b0, const C3<float> *b0_CYL, int n, 
    const float *r_GC, const C3<float> *curv_CYL, const C3<float> *grad_CYL, int nGC)
{

    int status = 0;
    C3<float> This_b0_CYL = kj_interp1D(r.c1, r_b0, b0_CYL, n, status);
    p.status = max(p.status, status);
#if DEBUG_EVAL_VGC >= 1
    if (status > 0) {
        cout << "ERROR 1 in eval_vGC" << endl;
        exit(1);
    }
#endif

    status = 0;
    C3<float> This_curv_CYL = kj_interp1D(r.c1, r_GC, curv_CYL, nGC, status);
    p.status = max(p.status, status);

#if DEBUG_EVAL_VGC >= 1
    if (status > 0) {
        cout << "ERROR 2 in eval_vGC" << endl;
        exit(1);
    }
#endif

    status = 0;
    C3<float> This_grad_CYL = kj_interp1D(r.c1, r_GC, grad_CYL, nGC, status);
    p.status = max(p.status, status);
#if DEBUG_EVAL_VGC >= 1
    if (status > 0) {
        cout << "ERROR 3 in eval_vGC" << endl;
        exit(1);
    }
#endif

#if DEBUG_EVAL_VGC >= 1

    cout << "r.c1: " << r.c1 << endl;
    cout << "p.c1: " << p.c1 << endl;
    cout << "vPar: " << vPar << endl;
    cout << "vPer: " << vPer << endl;
    cout << "b0_CYL: " << This_b0_CYL.c1 << "  " << This_b0_CYL.c2 << "  " << This_b0_CYL.c3 << endl;
    cout << "curv_CYL: " << This_curv_CYL.c1 << "  " << This_curv_CYL.c2 << "  " << This_curv_CYL.c3 << endl;
    cout << "grad_CYL: " << This_grad_CYL.c1 << "  " << This_grad_CYL.c2 << "  " << This_grad_CYL.c3 << endl
         << endl;
    cout << "max(grad_CYL): " << maxC3VecAbs(grad_CYL) << endl;

#endif

    C3<float> UnitB_CYL = This_b0_CYL / mag(This_b0_CYL);

    C3<float> vGC = vPar * UnitB_CYL + std::pow(vPer, 2) * This_grad_CYL + std::pow(vPar, 2) * This_curv_CYL;

    return vGC;
}

float GetAlpComp(const float vPer, const float phs)
{

    return vPer * sin(phs);
}

float GetBetComp(const float vPer, const float phs)
{

    return vPer * cos(phs);
}
